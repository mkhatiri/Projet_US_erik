#include "main-pr.hpp"

#define THROW_AWAY 0
#include "Padded2DArray.hpp"
#include <omp.h>
#include "memutils.hpp"
#include <cmath>

//#define SHOWLOADBALANCE
#include "logged_array.hpp"

//#define LOG
//#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <hipblas.h>
#include "hip/hip_runtime_api.h"
#include "math.h"
#include "tbb/concurrent_queue.h"
#include "math.h"
#include "AdaptativeUtils.hpp"

	template <typename VertexType, typename EdgeType, typename Scalar>
int main_pr(VertexType nVtx, EdgeType* xadj_, VertexType *adj_, Scalar* val_, Scalar *prior_, Scalar* pr_,
		Scalar lambda,
		int nTry, //algo parameter
		util::timestamp& totaltime, std::string& 
	   )
{

	int nb_blocks = 0;
	int blk_size = 0;
	int nb_threads = 0;
	int V = 0;

	{
		char* str = getenv ("NBTHREAD");
		if (str) {
			std::stringstream ss (str);
			ss>>nb_threads;
			if (!ss)
				std::cerr<<"NBTHREAD invalid"<<std::endl;
		}
	}



	{
		char* str = getenv ("BLKSIZE");
		if (str) {
			std::stringstream ss (str);
			ss>>blk_size;
			if (!ss)
				std::cerr<<"SUBSIZE invalid"<<std::endl;
		}
	}


	{
                char* str = getenv ("NBBLOCK");
                if (str) {
                        std::stringstream ss (str);
                        ss>>nb_blocks;
                        if (!ss)
                                std::cerr<<"NBBLOCK invalid"<<std::endl;
                }
        }

	{
                char* str = getenv ("VAL");
                if (str) {
                        std::stringstream ss (str);
                        ss>>V;
                        if (!ss)
                                std::cerr<<"val invalid"<<std::endl;
                }
        }



	if(nb_threads == 0 ){
		std::cerr<<" NBTHREAD=??? "<<std::endl;
		exit(0);
	}

	if(blk_size == 0 ){
                std::cerr<<" BLKSIZE=??? "<<std::endl;
                exit(0);
        }
	if(nb_blocks == 0 ){
                std::cerr<<" NBBLOCK=??? "<<std::endl;
                exit(0);
        }


	bool coldcache = true;

	util::timestamp start(0,0);

	//cpuside variables  
	Scalar* prin_ = new Scalar[nVtx];
	EdgeType* xadj = xadj_;
	VertexType *adj = adj_;
	Scalar* val = val_;
	Scalar* prior = prior_;
	Scalar* prin = prin_;
	Scalar* prout = pr_;
	Scalar alpha = lambda;
	Scalar beta = 1-lambda;


	//cuda side variable
	EdgeType* d_xadj ;
	VertexType *d_adj ;
	Scalar* d_val ;
	Scalar* d_prior ;
	Scalar* d_prin ;
	Scalar* d_prout ;
	Scalar *d_alpha;
	Scalar *d_beta;

	
	hipSetDevice(0);
	/* Get handle to the CUBLAS context */
	hipblasHandle_t cublasHandle = 0;
	hipblasStatus_t hipblasStatus_t;
	hipblasStatus_t = hipblasCreate(&cublasHandle);

	/* Get handle to the CUSPARSE context */
	hipsparseHandle_t cusparseHandle = 0;
	hipsparseStatus_t cusparseStatus;
	cusparseStatus = hipsparseCreate(&cusparseHandle);

	hipsparseMatDescr_t descr = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descr);

	hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);


	//memalloc

	checkCudaErrors( hipMalloc((void**)&d_xadj, (nVtx+1)*sizeof(*xadj)) );
	checkCudaErrors( hipMalloc((void**)&d_adj, (xadj[nVtx])*sizeof(*adj)) );
	checkCudaErrors( hipMalloc((void**)&d_val, (xadj[nVtx])*sizeof(*val)) );
	checkCudaErrors( hipMalloc((void**)&d_prior, (nVtx*sizeof(*prior))));
	checkCudaErrors( hipMalloc((void**)&d_prin, (nVtx*sizeof(*prin)) ));
	checkCudaErrors( hipMalloc((void**)&d_prout, (nVtx*sizeof(*prout)) ));

	//cpu to gpu copies

	checkCudaErrors( hipMemcpy(d_xadj, xadj, (nVtx+1)*sizeof(*xadj), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_adj, adj, (xadj[nVtx])*sizeof(*adj), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_val, val, (xadj[nVtx])*sizeof(*val), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_prior, prior, nVtx*sizeof(*prior), hipMemcpyHostToDevice) );



	int nRows = nVtx;
	unsigned long* rowBlocks;
	const int nThreadPerBlock = nb_threads; 
	const unsigned int blkSize = blk_size;
	const unsigned int blkMultiplier = 3 ;
	const unsigned int rows_for_vector = 1 ;
	const bool allocate_row_blocks = true;

	//device variable
	unsigned long* d_rowBlocks;
	unsigned int* d_blkSize;
	unsigned int* d_rows_for_vector;
	unsigned int* d_blkMultiplier;
	float* d_a;
	float* d_b;
	int rowBlockSize1;
	int rowBlockSize2;


//**************************************************
	int *method;
	int *d_method;
//	float *rowErr;

	method = (int*) calloc(sizeof(int), 5);
	checkCudaErrors( hipMalloc((void**)&d_method, 5*sizeof(int)));

	checkCudaErrors( hipMemcpy(d_method, method, 5*sizeof(int), hipMemcpyHostToDevice) );
//*********************************************


//	util::timestamp start_2(0,0);
//	start_2 = util::timestamp(); 
	unsigned long* rowBlockstest;

//	cout <<" comput rowBlocks " << endl;
	//calculer rowBlockSize
	rowBlockSize1 = ComputeRowBlocksSize<int,int>(xadj, nVtx, blkSize, blkMultiplier, rows_for_vector, nThreadPerBlock);
	//declarer rowBlocks

	//cerr << " rowBlockSize1=" << rowBlockSize1;
	
	rowBlockSize2 = rowBlockSize1;


	rowBlocks = (unsigned long*) calloc(sizeof(unsigned long),rowBlockSize1);
//	rowBlockstest = (unsigned long*) calloc(sizeof(unsigned long),rowBlockSize1);

	//calculer rowBlocks
	ComputeRowBlocks<int,int>( rowBlocks, rowBlockSize2, xadj, nVtx, blkSize, blkMultiplier, rows_for_vector, nThreadPerBlock, allocate_row_blocks);

//	cout << "fin de calcule de rowBlocks" <<endl;

//	cerr << "rowBlockSize2=" << rowBlockSize2 << endl;
//		if(rowBlocks[rowBlockSize1] == 0){
			//cout << "XrowBlocks[rowBlockSize1]=" <<  ((rowBlocks[rowBlockSize1 ] >> 32) & ((1UL << 32) - 1UL))  << endl;	
			//cout << "XrowBlocks[rowBlockSiz1e+1]=" <<  ((rowBlocks[rowBlockSize1+1 ] >> 32) & ((1UL << 32) - 1UL))  << endl;	
		//	rowBlockSize1--;
//		}else{

//			cout << " rowBlocks[rowBlockSize1]=" << rowBlocks[rowBlockSize1] << " rowBlocks[rowBlockSize1+1]=" << rowBlocks[rowBlockSize1+1]  <<  endl;
//			}
//

	//	util::timestamp stop_2;  
	//	util::timestamp	totaltime_2(0,0);
	//	totaltime_2 += stop_2 - start_2;
	//	char timestr[20];
	//	totaltime_2.to_c_str(timestr, 20);



	//malloc for device variable
//	checkCudaErrors( hipMalloc((void**)&rowErr, (rowBlockSize1+1)*sizeof(float)));
	checkCudaErrors( hipMalloc((void**)&d_rowBlocks, ((rowBlockSize1)*sizeof(unsigned long))));
	checkCudaErrors( hipMalloc((void**)&d_blkSize, 1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_rows_for_vector,1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_blkMultiplier, 1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_a, 1*sizeof(float)));
	checkCudaErrors( hipMalloc((void**)&d_b, 1*sizeof(float)));


	//send data to device
	checkCudaErrors( hipMemcpy(d_rowBlocks, rowBlocks, (rowBlockSize1)*sizeof(unsigned long), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_blkSize, &blkSize, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_rows_for_vector, &rows_for_vector, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_blkMultiplier, &blkMultiplier, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_a, &alpha, 1*sizeof(Scalar), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_b, &beta, 1*sizeof(Scalar), hipMemcpyHostToDevice) );

	int mmshared_size =  (blkSize + 1) * sizeof(float);


	// prepar stream config

	list<Task> *tasks = new  list<Task>;

	tbb::concurrent_bounded_queue<stream_container<int,int,float>* >* streams = new tbb::concurrent_bounded_queue<stream_container<int,int,float>* >;

//	int nb_blocks = 128;
	int stream_number = 1;
	

	int X, subsize;
	X = (int) rowBlockSize1/(nb_blocks) ;


	if(X > 32){
		if(X % 32 == 0){
			subsize = X;
		}else{  
			X = X / 32 ;
			subsize = (X+1) * 32;
		}
	}else{
		subsize=rowBlockSize1;
	}


	//	subsize = (int) (rowBlockSize1)/(nb_blocks) ;

	///	subsize = nb_blocks;
	//	if( rowBlockSize1/subsize > 4 )
	//		nb_blocks =  (int) (rowBlockSize1)/subsize;
	//	else
	//		{
	//			subsize = rowBlockSize1;
	//			nb_blocks =1;
	//		}


	cout << "nb_blocks=" << nb_blocks << " subsize=" << subsize << " rowBlockSize1=" << rowBlockSize1 << endl;



	//	while( X%2 != 0)
	//	{	nb_blocks++;
	//		X = (int) (rowBlockSize1)/(nb_blocks);
	//	}
	/*	
		while( subsize <= 10 )
		subsize = subsize*2 ;


		while(subsize >= 8000) 
		subsize = subsize/2;

	 */
	cerr << " subsize=" << subsize << " nb_blocks="<<nb_blocks << " " << "rowBlockSize1=" << rowBlockSize1 << " " ; 
	/*	if(rowBlockSize1 >= nb_blocks)
		{
		X = (int) rowBlockSize1/(nb_blocks) ;
		cerr << endl << "X = rowBlockSize1 = " << rowBlockSize1 << "/  nb_blocks = " <<  nb_blocks << " = " << X << endl ;
		}	
		else{
		X = (int) rowBlockSize1 / 4;
		cerr << endl << "X = rowBlockSize1 = " << rowBlockSize1 << "/  nb_blocks = " <<  nb_blocks << " = " << X << endl ;
		}
	 */
	/*if(X >=64)
	  {
	 ************************
	 if(X % 64 == 0){
	 subsize = X;  
	 cerr << "if -  subsize=" << subsize << endl ;
	 }else{
	 X = X / 64 ;
		subsize = (X+1) * 64;
		cerr << "else - subsize=" << subsize << endl ;
	}
****************
}else{
		subsize = X; 	
}
*************************/

/*
	int xadjPtr1 =  ((rowBlocks[rowBlockSize1] >> (64-32)) & ((1UL << 32) - 1UL));

	cout << "rowBlockSize : "<< rowBlockSize1 << " last row " << xadjPtr1 << endl;
	
	cout << "subsize : "<< subsize << endl;
	cout << "start creat stream " <<endl;
*/




	creat_stream<int, int, float>(d_rowBlocks, d_a, d_b, d_val, d_xadj, d_adj, d_prin, d_prout, d_blkSize, d_rows_for_vector, d_blkMultiplier, streams, stream_number );
	int nb_tasks = split_input_to_tasks(rowBlocks, rowBlockSize1, subsize, *tasks);
//	cout << "end creat stream " <<endl;
//
//	cout << "start split task " <<endl;
//	cout << "fin split task " << nb_tasks << endl;
//	int cum=0;
//	int S=0;
//	unsigned int total_row=0;

/*
	for(int tas=0; tas<nb_tasks; tas++){
		Task t = get_task(tasks, tas);
	//	cout << "task_id="<< t.id << " t.rowBlocksPtr=" << t.rowBlocksPtr << " t.rowBlockSize=" << t.rowBlockSize <<endl;
		cum=0;

		for(int Bid=t.rowBlocksPtr ; Bid< t.rowBlocksPtr + t.rowBlockSize; Bid++)
		{
			unsigned int row = ((rowBlocks[Bid] >> 32) & ((1UL << 32) - 1UL));	 // OWBITS = 32
			unsigned int stop_row = ((rowBlocks[Bid + 1] >> 32) & ((1UL << 32) - 1UL));
			unsigned int num_rows = stop_row - row;
			unsigned int wg = rowBlocks[Bid] & ((1 << 24) - 1);
	//		cout << cum  << " sum= "<< S << "row["<< Bid <<"]="<< row << " stop_row=" << stop_row << " num_row="  << num_rows << " Total_rows=" << total_row << " wg=" << wg << endl;

			total_row += num_rows;
			cum++;
		}

}*/


int m1=0, m2=0, m3=0;

for (int TRY=0; TRY<THROW_AWAY+nTry; ++TRY)
{
	//if (TRY >= THROW_AWAY)
	//	start = util::timestamp();

	for (int iter = 0; iter < 40; ++iter){
	
		int index =0;

		if (iter == 0)
			//std::copy (prior, prior+nVtx, prin);
			checkCudaErrors(hipMemcpy(d_prin, d_prior, nVtx*sizeof(*prior), hipMemcpyDeviceToDevice));
		else
			//std::copy (prout, prout+nVtx, prin);
			checkCudaErrors(hipMemcpy(d_prin, d_prout, nVtx*sizeof(*prout), hipMemcpyDeviceToDevice));

		Scalar eps = 0.;

		//prout = A prin
		//prout = lambda * prout + (1-lambda) prior

		checkCudaErrors(hipMemcpy(d_prout, d_prior, nVtx*sizeof(*prior), hipMemcpyDeviceToDevice));


		//for float it is S.
		//does prout = alpha A prin + beta prout

		while(index < nb_tasks){
				stream_container<int, int, float> *current_stream;
				Task t = get_task(tasks, index);
				streams->pop(current_stream);
				put_work_on_stream<int,int,float>(current_stream,t);
				//cudaPrintError("before kernel");


				csr_adaptative<<< current_stream->rowBlockSize , nThreadPerBlock, mmshared_size, current_stream->stream >>>(current_stream->d_val, current_stream->d_adj, current_stream->d_xadj, current_stream->d_prin, current_stream->d_prout, (current_stream->d_rowBlocks + current_stream->rowBlocksPtr), current_stream->alpha, current_stream->beta, current_stream->d_blkSize, current_stream->d_blkMultiplier, current_stream->d_rows_for_vector, current_stream->rowBlockSize, d_method);

				//	cudaPrintError("after kernel1");

				//	cudaPrintError("befor callback");
				hipStreamAddCallback(current_stream->stream, call_back , current_stream , 0);
				//	cudaPrintError("after callback");


				index++;
		}	


		//		cudaPrintError("befor Synch");
			//	hipDeviceSynchronize();
		//		cudaPrintError("after Synch");

		if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
			std::cerr<<"err-1"<<std::endl;



		//compute epsilon
		//using prin to compute epsilon
		float epsalpha = -1.;
	
		hipblasStatus_t = hipblasSaxpy (cublasHandle, nVtx, &epsalpha, d_prout, 1, d_prin, 1); // d_prin = d_prout*-1 + d_prin


		if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
			std::cerr<<"err-2"<<std::endl;

		start = util::timestamp();
		hipblasStatus_t = hipblasSasum(cublasHandle, nVtx, d_prin, 1, &eps);
		if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
			std::cerr<<"err-3"<<std::endl;


		util::timestamp stop2;  
		cout << "ad : totaltime = " << stop2 - start << endl;
		//	cudaPrintError("cublas");



		//stopping condition
		//	if (eps < 0) // deactivited for testing purposes
		//		iter = 20;

		std::cerr << eps << endl; 


	}
	checkCudaErrors(hipMemcpy(prout, d_prout, 1*sizeof(*prout), hipMemcpyDeviceToHost));

	//std::cerr<<"PR[0]="<<prout[0]<<std::endl;
	//	checkCudaErrors(hipMemcpy(method, d_method, 5*sizeof(int), hipMemcpyDeviceToHost));
	//	std::cerr << " method Stm="<< method[0] << " V ="<< method[1] << " VL="<< method[2] << " V3="<< method[3] << " V4=" <<method[5] <<endl ;

	//	for(int i=0; i<1; i++)
	//	{
	std::cerr.precision(10);
	std::cerr <<"PR["<< 0 <<"]="<<prout[0]<<std::endl;
	//	}
	if (TRY >= THROW_AWAY)
	{
		util::timestamp stop;  
		totaltime += stop - start;
		cout << "2cuda-adap : totaltime = " << stop - start << endl;
	}



	/*    
#ifndef LOG
if (coldcache) {
#pragma omp parallel
{
evict_array_from_cache(adj, xadj[nVtx]*sizeof(*adj));
evict_array_from_cache(xadj, (nVtx+1)*sizeof(*xadj));
evict_array_from_cache(val, xadj[nVtx]*sizeof(*val));
evict_array_from_cache(prior, nVtx*sizeof(*prior));
evict_array_from_cache(prin, nVtx*sizeof(*prin));
evict_array_from_cache(prout, nVtx*sizeof(*prout));

#pragma omp barrier
}
}
#endif
	 */
}


//cudaPrintError(" hipDeviceReset() -1 ");
//hipDeviceReset();
//cudaPrintError(" hipDeviceReset() -2 ");
hipFree(d_rowBlocks);
hipFree(d_blkSize);
hipFree(d_rows_for_vector);
hipFree(d_blkMultiplier);
hipFree(d_a);
hipFree(d_b);

free(rowBlocks);
free(method);
free(rowBlockstest);

#ifdef SHOWLOADBALANCE
std::cout<<"load balance"<<std::endl;
for (int i=0; i< 244; ++i)
std::cout<<count[i]<<std::endl;
#endif

//delete[] prin_;



return 0;
}



