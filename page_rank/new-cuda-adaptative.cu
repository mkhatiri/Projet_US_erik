#include "main-pr.hpp"

#define THROW_AWAY 0
#include "Padded2DArray.hpp"
#include <omp.h>
#include "memutils.hpp"
#include <cmath>

//#define SHOWLOADBALANCE
#include "logged_array.hpp"

//#define LOG
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <hipblas.h>
#include "hip/hip_runtime_api.h"
#include "math.h"
#include "tbb/concurrent_queue.h"
#include "math.h"
#include "AdaptativeUtils.hpp"

	template <typename VertexType, typename EdgeType, typename Scalar>
int main_pr(VertexType nVtx, EdgeType* xadj_, VertexType *adj_, Scalar* val_, Scalar *prior_, Scalar* pr_,
		Scalar lambda,
		int nTry, //algo parameter
		util::timestamp& totaltime, std::string& 
	   )
{

	int nb_blocks = 0;
	int blk_size = 0;
	int nb_threads = 0;
	int V = 0;

	{
		char* str = getenv ("NBTHREAD");
		if (str) {
			std::stringstream ss (str);
			ss>>nb_threads;
			if (!ss)
				std::cerr<<"NBTHREAD invalid"<<std::endl;
		}
	}



	{
		char* str = getenv ("BLKSIZE");
		if (str) {
			std::stringstream ss (str);
			ss>>blk_size;
			if (!ss)
				std::cerr<<"SUBSIZE invalid"<<std::endl;
		}
	}


	{
                char* str = getenv ("NBBLOCK");
                if (str) {
                        std::stringstream ss (str);
                        ss>>nb_blocks;
                        if (!ss)
                                std::cerr<<"NBBLOCK invalid"<<std::endl;
                }
        }

	{
                char* str = getenv ("VAL");
                if (str) {
                        std::stringstream ss (str);
                        ss>>V;
                        if (!ss)
                                std::cerr<<"val invalid"<<std::endl;
                }
        }



	if(nb_threads == 0 ){
		std::cerr<<" NBTHREAD=??? "<<std::endl;
		exit(0);
	}

	if(blk_size == 0 ){
                std::cerr<<" BLKSIZE=??? "<<std::endl;
                exit(0);
        }
	if(nb_blocks == 0 ){
                std::cerr<<" NBBLOCK=??? "<<std::endl;
                exit(0);
        }


	bool coldcache = true;

	util::timestamp start(0,0);

	//cpuside variables  
	Scalar* prin_ = new Scalar[nVtx];
	EdgeType* xadj = xadj_;
	VertexType *adj = adj_;
	Scalar* val = val_;
	Scalar* prior = prior_;
	Scalar* prin = prin_;
	Scalar* prout = pr_;
	Scalar alpha = lambda;
	Scalar beta = 1-lambda;


	//cuda side variable
	EdgeType* d_xadj ;
	VertexType *d_adj ;
	Scalar* d_val ;
	Scalar* d_prior ;
	Scalar* d_prin ;
	Scalar* d_prout ;
	Scalar *d_alpha;
	Scalar *d_beta;

	
	hipSetDevice(0);
	/* Get handle to the CUBLAS context */
	hipblasHandle_t cublasHandle = 0;
	hipblasStatus_t hipblasStatus_t;
	hipblasStatus_t = hipblasCreate(&cublasHandle);

	/* Get handle to the CUSPARSE context */
	hipsparseHandle_t cusparseHandle = 0;
	hipsparseStatus_t cusparseStatus;
	cusparseStatus = hipsparseCreate(&cusparseHandle);

	hipsparseMatDescr_t descr = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descr);

	hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);


	//memalloc

	checkCudaErrors( hipMalloc((void**)&d_xadj, (nVtx+1)*sizeof(*xadj)) );
	checkCudaErrors( hipMalloc((void**)&d_adj, (xadj[nVtx])*sizeof(*adj)) );
	checkCudaErrors( hipMalloc((void**)&d_val, (xadj[nVtx])*sizeof(*val)) );
	checkCudaErrors( hipMalloc((void**)&d_prior, (nVtx*sizeof(*prior))));
	checkCudaErrors( hipMalloc((void**)&d_prin, (nVtx*sizeof(*prin)) ));
	checkCudaErrors( hipMalloc((void**)&d_prout, (nVtx*sizeof(*prout)) ));

	//cpu to gpu copies

	checkCudaErrors( hipMemcpy(d_xadj, xadj, (nVtx+1)*sizeof(*xadj), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_adj, adj, (xadj[nVtx])*sizeof(*adj), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_val, val, (xadj[nVtx])*sizeof(*val), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_prior, prior, nVtx*sizeof(*prior), hipMemcpyHostToDevice) );



	int nRows = nVtx;
	unsigned long* rowBlocks;
	const int nThreadPerBlock = nb_threads; 
	const unsigned int blkSize = blk_size;
	const unsigned int blkMultiplier = 3 ;
	const unsigned int rows_for_vector = 8 ;
	const bool allocate_row_blocks = true;

	//device variable
	unsigned long* d_rowBlocks;
	unsigned int* d_blkSize;
	unsigned int* d_rows_for_vector;
	unsigned int* d_blkMultiplier;
	float* d_a;
	float* d_b;
	int rowBlockSize1;
	int rowBlockSize2;

	int *rBS;

//**************************************************
	int *method;
	int *d_method;
	float *rowErr;

	method = (int*) calloc(sizeof(int), 5);
	checkCudaErrors( hipMalloc((void**)&d_method, 5*sizeof(int)));
	checkCudaErrors( hipMalloc((void**)&d_method, blkSize*sizeof(float)));

	checkCudaErrors( hipMemcpy(d_method, method, 5*sizeof(int), hipMemcpyHostToDevice) );
//*********************************************


	unsigned long* rowBlockstest;

	cout <<" comput rowBlocks " << endl;
	//calculer rowBlockSize
	rowBlockSize1 = ComputeRowBlocksSize<int,int>(xadj, nVtx, blkSize, blkMultiplier, rows_for_vector, nThreadPerBlock);
	//declarer rowBlocks

	cerr << " rowBlockSize1=" << rowBlockSize1;
	
	rowBlockSize2 = rowBlockSize1;


	rowBlocks = (unsigned long*) calloc(sizeof(unsigned long),rowBlockSize1);
	rowBlockstest = (unsigned long*) calloc(sizeof(unsigned long),rowBlockSize1);
	rBS = (int*) calloc(sizeof(int*),1);

	//calculer rowBlocks
	ComputeRowBlocks<int,int>( rowBlocks, rowBlockSize2, xadj, nVtx, blkSize, blkMultiplier, rows_for_vector, nThreadPerBlock, allocate_row_blocks);

	cout << "fin de calcule de rowBlocks" <<endl;


//	cerr << "rowBlockSize2=" << rowBlockSize2 << endl;



	//	if(rowBlocks[rowBlockSize1] == 0){
	//		rowBlockSize1--;
	//	}


	//malloc for device variable
	checkCudaErrors( hipMalloc((void**)&d_rowBlocks, ((rowBlockSize1)*sizeof(unsigned long))));
	checkCudaErrors( hipMalloc((void**)&d_blkSize, 1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_rows_for_vector,1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_blkMultiplier, 1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_a, 1*sizeof(float)));
	checkCudaErrors( hipMalloc((void**)&d_b, 1*sizeof(float)));


	//send data to device
	checkCudaErrors( hipMemcpy(d_rowBlocks, rowBlocks, (rowBlockSize1)*sizeof(unsigned long), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_blkSize, &blkSize, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_rows_for_vector, &rows_for_vector, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_blkMultiplier, &blkMultiplier, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_a, &alpha, 1*sizeof(Scalar), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_b, &beta, 1*sizeof(Scalar), hipMemcpyHostToDevice) );

	int mmshared_size =  (blkSize + 1) * sizeof(float);


	// prepar stream config

	list<Task> *tasks = new  list<Task>;

	tbb::concurrent_bounded_queue<stream_container<int,int,float>* >* streams = new tbb::concurrent_bounded_queue<stream_container<int,int,float>* >;

//	int nb_blocks = 128;
	int stream_number = 1;
	

	int X, subsize;


	X = (int) (rowBlockSize1)/(nb_blocks) ;
	
//	while( X%2 != 0)
//	{	nb_blocks++;
//		X = (int) (rowBlockSize1)/(nb_blocks);
//	}
	/*	
		while( subsize <= 10 )
		subsize = subsize*2 ;


		while(subsize >= 8000) 
		subsize = subsize/2;

	 */
	subsize = X;
	cerr << " subsize=" << subsize << " nb_blocks="<<nb_blocks << " "  ; 
	/*	if(rowBlockSize1 >= nb_blocks)
		{
		X = (int) rowBlockSize1/(nb_blocks) ;
		cerr << endl << "X = rowBlockSize1 = " << rowBlockSize1 << "/  nb_blocks = " <<  nb_blocks << " = " << X << endl ;
		}	
		else{
		X = (int) rowBlockSize1 / 4;
		cerr << endl << "X = rowBlockSize1 = " << rowBlockSize1 << "/  nb_blocks = " <<  nb_blocks << " = " << X << endl ;
		}
	 */
	/*if(X >=64)
	  {
	 ************************
	 if(X % 64 == 0){
	 subsize = X;  
	 cerr << "if -  subsize=" << subsize << endl ;
	 }else{
	 X = X / 64 ;
		subsize = (X+1) * 64;
		cerr << "else - subsize=" << subsize << endl ;
	}
****************
}else{
		subsize = X; 	
}
*************************/


	int xadjPtr1 =  ((rowBlocks[rowBlockSize1] >> (64-32)) & ((1UL << 32) - 1UL));

	cout << "rowBlockSize : "<< rowBlockSize1 << " last row " << xadjPtr1 << endl;
	
	cout << "subsize : "<< subsize << endl;
	cout << "start creat stream " <<endl;


	creat_stream<int, int, float>(d_rowBlocks, d_a, d_b, d_val, d_xadj, d_adj, d_prin, d_prout, d_blkSize, d_rows_for_vector, d_blkMultiplier, streams, stream_number );
	cout << "end creat stream " <<endl;

	rowBlockSize1--;
	cout << "start split task " <<endl;
	int nb_tasks = split_input_to_tasks(rowBlocks, rowBlockSize1, subsize, *tasks);
	cout << "fin split task " <<endl;
int cum=0;
int S=0;
unsigned int total_row=0;
for(int nbT=0; nbT< nb_tasks; nbT++){

	Task t = get_task(tasks, nbT);
	cout << "task_id="<< t.id << " t.rowBlocksPtr=" << t.rowBlocksPtr << " t.rowBlockSize=" << t.rowBlockSize <<endl;
	S += t.rowBlockSize;
	cum=0;
	
	for(int Bid=t.rowBlocksPtr ; Bid<= S; Bid++)
		{
		unsigned int row = ((rowBlocks[Bid] >> 32) & ((1UL << 32) - 1UL));	 // OWBITS = 32
		unsigned int stop_row = ((rowBlocks[Bid + 1] >> 32) & ((1UL << 32) - 1UL));
		unsigned int num_rows = stop_row - row;
		total_row += num_rows;
		cout << cum  << " sum= "<< S << "row["<< Bid <<"]="<< row << " stop_row=" << stop_row << " num_row="  << num_rows << " Total_rows=" << total_row << endl;
		cum++;
		}
}

//	for(int i=0; i<nb_tasks; i++){
//		Task t = get_task(tasks, i);
//		cerr << "id : " << t.id <<" - rowBlocksPtr " << t.rowBlocksPtr <<" - rowBlockSize " << t.rowBlockSize <<endl;

//	}



	cout << "task lengh = " << tasks->size() << endl ;

	int m1=0, m2=0, m3=0;

	for (int TRY=0; TRY<THROW_AWAY+nTry; ++TRY)
	{
		if (TRY >= THROW_AWAY)
			start = util::timestamp();

		for (int iter = 0; iter < 1 ; ++ iter) {

			int index = 0;
			//setup prin
			if (iter == 0)
				//std::copy (prior, prior+nVtx, prin);
				checkCudaErrors(hipMemcpy(d_prin, d_prior, nVtx*sizeof(*prior), hipMemcpyDeviceToDevice));
			else
				//std::copy (prout, prout+nVtx, prin);
				checkCudaErrors(hipMemcpy(d_prin, d_prout, nVtx*sizeof(*prout), hipMemcpyDeviceToDevice));

			Scalar eps = 0.;

			//prout = A prin
			//prout = lambda * prout + (1-lambda) prior

			checkCudaErrors(hipMemcpy(d_prout, d_prior, nVtx*sizeof(*prior), hipMemcpyDeviceToDevice));


			//for float it is S.
			//does prout = alpha A prin + beta prout

			while(index == 0 ){ //< nb_tasks){
				stream_container<int, int, float> *current_stream;
				Task t = get_task(tasks, index);
				streams->pop(current_stream);
				put_work_on_stream<int,int,float>(current_stream,t);
				cudaPrintError("before kernel");
				
				 cout << "index" << index << " rowBlockSize" << current_stream->rowBlockSize  << endl;
			//	 cerr << "index" << index << endl;


//checkCudaErrors(hipMemcpy(rowBlockstest, (current_stream->d_rowBlocks+current_stream->rowBlocksPtr),(current_stream->rowBlockSize+1)*sizeof(unsigned long), hipMemcpyDeviceToHost));
	//	cerr << "=current_stream->rowBlockSize " << current_stream->rowBlockSize  << endl;		
for(int i=0;i<=current_stream->rowBlockSize;i++)
{
	
	int x =  ((rowBlockstest[i] >> (64-32)) & ((1UL << 32) - 1UL));
//	std::cerr<< "i=" << i << " current_stream->rowBlockSize=" << current_stream->rowBlockSize  << " rowblokstest[" <<i <<"]="<< x <<std::endl;

}
//	cout <<" fin .......... " <<endl;

				//cerr << index << " -> task id = " << t.id << "current_stream->rowBlockSize=" << current_stream->rowBlockSize << " rowBlocksPtr=" << t.rowBlocksPtr << " subsize=" << subsize << "current_stream->rowBlocksPtr " << current_stream->rowBlocksPtr ;
				//cerr << " nTheeadPerBlock=" << nThreadPerBlock  << " mmshared_size=" << mmshared_size << endl ;
	
				csr_adaptative<<< current_stream->rowBlockSize , nThreadPerBlock, mmshared_size, current_stream->stream >>>(current_stream->d_val, current_stream->d_adj, current_stream->d_xadj, current_stream->d_prin, current_stream->d_prout, (current_stream->d_rowBlocks + current_stream->rowBlocksPtr), current_stream->alpha, current_stream->beta, current_stream->d_blkSize, current_stream->d_blkMultiplier, current_stream->d_rows_for_vector, current_stream->rowBlockSize, d_method, rowErr);

                                cudaPrintError("after kernel1");
//
//				checkCudaErrors(hipMemcpy(method, d_method, 3*sizeof(int), hipMemcpyDeviceToHost));
//				std::cerr << index << " method Stm="<< method[0]-m1 << " V ="<< method[1]-m2 << " VL="<< method[2]-m3 << endl;
				

                                cudaPrintError("befor callback");
				hipStreamAddCallback(current_stream->stream, call_back , current_stream , 0);
                                cudaPrintError("after callback");






				index++;
			}

                                cudaPrintError("befor Synch");
				hipDeviceSynchronize();
                                cudaPrintError("after Synch");

			if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
				std::cerr<<"err-1"<<std::endl;



			//compute epsilon
			//using prin to compute epsilon
			float epsalpha = -1.;
			hipblasStatus_t = hipblasSaxpy (cublasHandle, nVtx, &epsalpha, d_prout, 1, d_prin, 1); // d_prin = d_prout*-1 + d_prin

			if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
				std::cerr<<"err-2"<<std::endl;

			hipblasStatus_t = hipblasSasum(cublasHandle, nVtx, d_prin, 1, &eps);
			if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
				std::cerr<<"err-3"<<std::endl;


                                cudaPrintError("cublas");



			//stopping condition
		//	if (eps < 0) // deactivited for testing purposes
		//		iter = 20;

			std::cerr<< " VAL=" << V << " -> "<< eps; 

		}

		checkCudaErrors(hipMemcpy(prout, d_prout, nVtx*sizeof(*prout), hipMemcpyDeviceToHost));

//		std::cerr<<"PR[0]="<<prout[0]<<std::endl;
				checkCudaErrors(hipMemcpy(method, d_method, 5*sizeof(int), hipMemcpyDeviceToHost));
				std::cerr << " method Stm="<< method[0] << " V ="<< method[1] << " VL="<< method[2] << " V3="<< method[3] << " V4=" <<method[5] <<endl;

for(int i=0; i<nVtx; i++)
     std::cerr<<"PR["<< i <<"]="<<prout[i]<<std::endl;

		if (TRY >= THROW_AWAY)
		{
			util::timestamp stop;  
			totaltime += stop - start;
		}


		/*    
#ifndef LOG
if (coldcache) {
#pragma omp parallel
{
evict_array_from_cache(adj, xadj[nVtx]*sizeof(*adj));
evict_array_from_cache(xadj, (nVtx+1)*sizeof(*xadj));
evict_array_from_cache(val, xadj[nVtx]*sizeof(*val));
evict_array_from_cache(prior, nVtx*sizeof(*prior));
evict_array_from_cache(prin, nVtx*sizeof(*prin));
evict_array_from_cache(prout, nVtx*sizeof(*prout));

#pragma omp barrier
}
}
#endif
	 */
}


				cudaPrintError(" hipDeviceReset() -1 ");
	hipDeviceReset();
				cudaPrintError(" hipDeviceReset() -2 ");
hipFree(d_rowBlocks);
hipFree(d_blkSize);
hipFree(d_rows_for_vector);
hipFree(d_blkMultiplier);
hipFree(d_a);
hipFree(d_b);

free(rowBlocks);
free(method);
free(rowBlockstest);
free(rBS);

#ifdef SHOWLOADBALANCE
std::cout<<"load balance"<<std::endl;
for (int i=0; i< 244; ++i)
std::cout<<count[i]<<std::endl;
#endif

//delete[] prin_;



return 0;
}



