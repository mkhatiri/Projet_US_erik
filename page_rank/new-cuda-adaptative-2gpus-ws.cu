#include "main-pr.hpp"

#define THROW_AWAY 1
#include "Padded2DArray.hpp"
#include <omp.h>
#include "memutils.hpp"
#include <cmath>

//#define SHOWLOADBALANCE
#include "logged_array.hpp"

//#define LOG
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <hipblas.h>
#include "hip/hip_runtime_api.h"
#include "math.h"
//#include "streamUtils.hpp"
#include "tbb/concurrent_queue.h"
#include "AdaptativeUtils.hpp"


	template <typename VertexType, typename EdgeType, typename Scalar>
int main_pr(VertexType nVtx, EdgeType* xadj_, VertexType *adj_, Scalar* val_, Scalar *prior_, Scalar* pr_,
		Scalar lambda,
		int nTry, //algo parameter
		util::timestamp& totaltime, std::string& out 
	   )
{
	int nb_blocks = 0;
	int blk_size = 0;
	int nb_threads = 0;
	int WGSIZE = 256;

	{
		char* str = getenv ("NBTHREAD");
		if (str) {
			std::stringstream ss (str);
			ss>>nb_threads;
			if (!ss)
				std::cerr<<"NBTHREAD invalid"<<std::endl;
		}
	}


	{
		char* str = getenv ("NBBLOCK");
		if (str) {
			std::stringstream ss (str);
			ss>>nb_blocks;
			if (!ss)
				std::cerr<<"NBBLOCK invalid"<<std::endl;
		}
	}




	{
		char* str = getenv ("BLKSIZE");
		if (str) {
			std::stringstream ss (str);
			ss>>blk_size;
			if (!ss)
				std::cerr<<"SUBSIZE invalid"<<std::endl;
		}
	}

	{
                char* str = getenv ("WGSIZE");
                if (str) {
                        std::stringstream ss (str);
                        ss>>WGSIZE;
                        if (!ss)
                                std::cerr<<"val invalid"<<std::endl;
                }
        }


	if(nb_threads == 0 ){
		std::cerr<<" NBTHREAD=??? "<<std::endl;
		exit(0);
	}

	if(blk_size == 0 ){
		std::cerr<<" BLKSIZE=??? "<<std::endl;
		exit(0);  
	}
	if(nb_blocks == 0 ){
		std::cerr<<" NBBLOCK=??? "<<std::endl;
		exit(0);
	}

	bool coldcache = true;

	util::timestamp start(0,0);

	//cpuside variables  
	Scalar* prin_ = new Scalar[nVtx];
	EdgeType* xadj = xadj_;
	VertexType *adj = adj_;
	Scalar* val = val_;
	Scalar* prior = prior_;
	Scalar* prin = prin_;
	Scalar* prout = pr_;
	Scalar alpha = lambda;
	Scalar beta = 1-lambda;

	Scalar alpha1 = lambda;
	Scalar beta1 = 1-lambda;
	Scalar epsalpha = -1;

	Scalar *h_eps0;
	Scalar *h_eps1;


	//cuda side variable
	EdgeType* d_xadj0 ;
	VertexType *d_adj0 ;
	Scalar* d_val0 ;
	Scalar* d_prior0 ;
	Scalar* d_prin0 ;
	Scalar* d_prout0 ;
	Scalar *d_alpha0;
	Scalar *d_beta0;
	Scalar *d_epsalpha0;
	Scalar *d_eps0;

	EdgeType* d_xadj1 ;
	VertexType *d_adj1 ;
	Scalar* d_val1 ;
	Scalar* d_prior1 ;
	Scalar* d_prin1 ;
	Scalar* d_prout1 ;
	Scalar *d_alpha1;
	Scalar *d_beta1;

	Scalar *d_epsalpha1;
	Scalar *d_eps1;

	/* Get handle to the CUBLAS context */
	hipSetDevice(0);
	hipblasHandle_t cublasHandle0 = 0;
	hipblasStatus_t cublasStatus0;
	cublasStatus0 = hipblasCreate(&cublasHandle0);
	hipblasSetPointerMode(cublasHandle0, HIPBLAS_POINTER_MODE_DEVICE);

	checkCudaErrors( hipSetDevice(1));
	hipblasHandle_t cublasHandle1 = 0;
	hipblasStatus_t cublasStatus1;
	cublasStatus1 = hipblasCreate(&cublasHandle1);
	hipblasSetPointerMode(cublasHandle1, HIPBLAS_POINTER_MODE_DEVICE);

	/* Get handle to the CUSPARSE context */
	hipSetDevice(0);

	hipsparseHandle_t cusparseHandle0 = 0;
	hipsparseStatus_t cusparseStatus0;
	cusparseStatus0 = hipsparseCreate(&cusparseHandle0);

	hipsparseMatDescr_t descr0 = 0;
	cusparseStatus0 = hipsparseCreateMatDescr(&descr0);

	hipsparseSetMatType(descr0,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr0,HIPSPARSE_INDEX_BASE_ZERO);


	hipSetDevice(1);

	hipsparseHandle_t cusparseHandle1 = 0;
	hipsparseStatus_t cusparseStatus1;
	cusparseStatus1 = hipsparseCreate(&cusparseHandle1);

	hipsparseMatDescr_t descr1 = 0;
	cusparseStatus1 = hipsparseCreateMatDescr(&descr1);

	hipsparseSetMatType(descr1,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr1,HIPSPARSE_INDEX_BASE_ZERO);


	//cuda stream

	hipSetDevice(0);

	hipStream_t stream0;
	hipStreamCreate(&stream0);

	hipSetDevice(1);

	hipStream_t stream1;
	hipStreamCreate(&stream1);

	//cuda variable 

	hipSetDevice(0);

	int *d_end; 		
	checkCudaErrors( hipMalloc((void**)&d_end, sizeof(*d_end)) );
	hipDeviceEnablePeerAccess(1, 0);

	hipSetDevice(1);

	int *d_begin; 		
	checkCudaErrors( hipMalloc((void**)&d_begin, sizeof(*d_begin)) );
	hipDeviceEnablePeerAccess(0, 0);
	//memalloc

	hipSetDevice(0);

	checkCudaErrors( hipMalloc((void**)&d_xadj0, (nVtx+1)*sizeof(*xadj)) );
	checkCudaErrors( hipMalloc((void**)&d_adj0, (xadj[nVtx])*sizeof(*adj)) );
	checkCudaErrors( hipMalloc((void**)&d_val0, (xadj[nVtx])*sizeof(*val)) );
	checkCudaErrors( hipMalloc((void**)&d_prior0, (nVtx*sizeof(*prior))));
	checkCudaErrors( hipMalloc((void**)&d_prin0, (nVtx*sizeof(*prin)) ));
	checkCudaErrors( hipMalloc((void**)&d_prout0, (nVtx*sizeof(*prout)) ));
	checkCudaErrors( hipMalloc((void**)&d_epsalpha0, (sizeof(epsalpha)) ));
	checkCudaErrors( hipHostMalloc((void**)&h_eps0, (sizeof(*h_eps0)) ));
	checkCudaErrors( hipMalloc((void**)&d_eps0, (sizeof(*h_eps0)) ));


	hipSetDevice(1);

	checkCudaErrors( hipMalloc((void**)&d_xadj1, (nVtx+1)*sizeof(*xadj)) );
	checkCudaErrors( hipMalloc((void**)&d_adj1, (xadj[nVtx])*sizeof(*adj)) );
	checkCudaErrors( hipMalloc((void**)&d_val1, (xadj[nVtx])*sizeof(*val)) );
	checkCudaErrors( hipMalloc((void**)&d_prior1, (nVtx*sizeof(*prior))));
	checkCudaErrors( hipMalloc((void**)&d_prin1, (nVtx*sizeof(*prin)) ));
	checkCudaErrors( hipMalloc((void**)&d_prout1, (nVtx*sizeof(*prout)) ));
	checkCudaErrors( hipMalloc((void**)&d_epsalpha1, (sizeof(epsalpha)) ));
	checkCudaErrors( hipHostMalloc((void**)&h_eps1, (sizeof(*h_eps1)) ));
	checkCudaErrors( hipMalloc((void**)&d_eps1, (sizeof(*h_eps1)) ));


	//cpu to gpu copies

	hipSetDevice(0);


	checkCudaErrors( hipMemcpy(d_xadj0, xadj, (nVtx+1)*sizeof(*xadj), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_adj0, adj, (xadj[nVtx])*sizeof(*adj), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_val0, val, (xadj[nVtx])*sizeof(*val), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_prior0, prior, nVtx*sizeof(*prior), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_epsalpha0, &epsalpha, sizeof(epsalpha), hipMemcpyHostToDevice) );


	hipSetDevice(1);


	checkCudaErrors( hipMemcpy(d_xadj1, xadj, (nVtx+1)*sizeof(*xadj), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_adj1, adj, (xadj[nVtx])*sizeof(*adj), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_val1, val, (xadj[nVtx])*sizeof(*val), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_prior1, prior, nVtx*sizeof(*prior), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_epsalpha1, &epsalpha, sizeof(epsalpha), hipMemcpyHostToDevice) );



	//tester PeerAcces

	

	int nRows = nVtx;
	unsigned long* rowBlocks;
	const int nThreadPerBlock = nb_threads; 
	const unsigned int blkSize = blk_size; 
	const unsigned int blkMultiplier = 3;
	const unsigned int rows_for_vector = 2; 
	const bool allocate_row_blocks = true;

	//device 0 variable 
	unsigned long* d_rowBlocks0;
	unsigned int* d_blkSize0;
	unsigned int* d_rows_for_vector0;
	unsigned int* d_blkMultiplier0;
	float* d_a0;
	float* d_b0;

	//device 1 variable 
	unsigned long* d_rowBlocks1;
	unsigned int* d_blkSize1;
	unsigned int* d_rows_for_vector1;
	unsigned int* d_blkMultiplier1;
	float* d_a1;
	float* d_b1;

	int rowBlockSize1;
	int rowBlockSize2;


	//calculer rowBlockSize
	rowBlockSize1 = ComputeRowBlocksSize<int,int>(xadj, nVtx, blkSize, blkMultiplier, rows_for_vector, nThreadPerBlock);
	//cout << "rowBlockSize1 : " << rowBlockSize1 << endl;

	//declarer rowBlocks
	rowBlocks = (unsigned long*) calloc(sizeof(unsigned long),rowBlockSize1);
	rowBlockSize2 = rowBlockSize1;

	//calculer rowBlocks
	ComputeRowBlocks<int,int>( rowBlocks, rowBlockSize2, xadj, nVtx, blkSize, blkMultiplier, rows_for_vector, nThreadPerBlock, allocate_row_blocks);
	//cout << "rowBlockSize2 : " << rowBlockSize2 <<endl;

//	int end = ((rowBlocks[rowBlockSize1] >> (64-32)) & ((1UL << 32) - 1UL));
//	cout << " end : " << end <<endl;

	cout << "rowBlockSize1 : " << rowBlockSize1 << endl;

	//malloc for device 0 variable
	hipSetDevice(0);
	checkCudaErrors( hipMalloc((void**)&d_rowBlocks0, (rowBlockSize1*sizeof(unsigned long))));
	checkCudaErrors( hipMalloc((void**)&d_blkSize0, 1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_rows_for_vector0,1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_blkMultiplier0, 1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_a0, 1*sizeof(float)));
	checkCudaErrors( hipMalloc((void**)&d_b0, 1*sizeof(float)));

	//malloc for device 1 variable
	hipSetDevice(1);
	checkCudaErrors( hipMalloc((void**)&d_rowBlocks1, (rowBlockSize1*sizeof(unsigned long))));
	checkCudaErrors( hipMalloc((void**)&d_blkSize1, 1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_rows_for_vector1,1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_blkMultiplier1, 1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_a1, 1*sizeof(float)));
	checkCudaErrors( hipMalloc((void**)&d_b1, 1*sizeof(float)));




	//send data to device 0 
	hipSetDevice(0);
	checkCudaErrors( hipMemcpy(d_rowBlocks0, rowBlocks, rowBlockSize1*sizeof(unsigned long), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_blkSize0, &blkSize, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_rows_for_vector0, &rows_for_vector, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_blkMultiplier0, &blkMultiplier, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_a0, &alpha, 1*sizeof(Scalar), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_b0, &beta, 1*sizeof(Scalar), hipMemcpyHostToDevice) );

	//send data to device 1 
	hipSetDevice(1);
	checkCudaErrors( hipMemcpy(d_rowBlocks1, rowBlocks, rowBlockSize1*sizeof(unsigned long), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_blkSize1, &blkSize, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_rows_for_vector1, &rows_for_vector, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_blkMultiplier1, &blkMultiplier, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_a1, &alpha, 1*sizeof(Scalar), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_b1, &beta, 1*sizeof(Scalar), hipMemcpyHostToDevice) );


	// prepar stream config

	list<Task> *tasks = new  list<Task>;

	tbb::concurrent_bounded_queue<stream_container<int,int,float>* >* streams = new tbb::concurrent_bounded_queue<stream_container<int,int,float>* >;

	int mmshared_size =  (blkSize + 1) * sizeof(float);

	// int nb_blocks = 512;
	int stream_number = 2;


	int X, subsize;
	X = (int) rowBlockSize1/(nb_blocks) ;

	if(X > 32){
		if(X % 32 == 0){
			subsize = X;
		}else{  
			X = X / 32 ;
			subsize = (X+1) * 32;
		}
	}else{
		if(rowBlockSize1%2==0)	subsize=rowBlockSize1/2; else  subsize=(rowBlockSize1+1)/2;
	}

	cout << "nb_blocks=" << nb_blocks << " subsize=" << subsize << " rowBlockSize1=" << rowBlockSize1 << endl;


	cout << "start creat stream " <<endl;
	creat_stream_2gpus<int, int, float>(d_rowBlocks0, d_a0, d_b0, d_val0, d_xadj0, d_adj0, d_prin0, d_prout0, d_blkSize0, d_rows_for_vector0, d_blkMultiplier0, d_rowBlocks1, d_a1, d_b1, d_val1, d_xadj1, d_adj1, d_prin1, d_prout1, d_blkSize1, d_rows_for_vector1, d_blkMultiplier1, streams, stream_number );

	cout << "end creat stream " <<endl;

	cout << "start split task " <<endl;
	int nb_tasks = split_input_to_tasks(rowBlocks, rowBlockSize1, subsize, *tasks);
	cout << "fin split task " << "nb_tasks=" << nb_tasks << endl;

//	nb_tasks--;


	int begin = rowBlockSize1;
	int end = 0;

	hipSetDevice(0);
	checkCudaErrors( hipMemcpy(d_end, &end, sizeof(*d_end), hipMemcpyHostToDevice) );

	hipSetDevice(1);
	checkCudaErrors( hipMemcpy(d_begin, &begin, sizeof(*d_begin), hipMemcpyHostToDevice) );





	int size =  (blkSize) * sizeof(float);

	int medium;
	for (int TRY=0; TRY<THROW_AWAY+nTry; ++TRY)
	{
		if (TRY >= THROW_AWAY)
			start = util::timestamp();

		int maxiter = 1;
	
		//medium =  ((rowBlocks[mediumRowblocks] >> (64-32)) & ((1UL << 32) - 1UL));
		//for GPU0

		hipSetDevice(0);
		//setup prin
		hipMemcpyAsync(d_prin0, d_prior0, nVtx*sizeof(*prior), hipMemcpyDeviceToDevice,stream0);
		//hipMemcpyAsync(d_prin0, d_prior0, (medium)*sizeof(*prior), hipMemcpyDeviceToDevice,stream0);


		hipSetDevice(1);
		//setup prin
		hipMemcpyAsync(d_prin1, d_prior1, nVtx*sizeof(*prior), hipMemcpyDeviceToDevice,stream1);

		//	hipMemcpyAsync(d_prin1+medium, d_prior1+medium, (nVtx-medium)*sizeof(*prior), hipMemcpyDeviceToDevice,stream1);


		hipSetDevice(1);
		checkCudaErrors( hipStreamSynchronize(stream1));


		hipSetDevice(0);
		checkCudaErrors( hipStreamSynchronize(stream0));

		begin = rowBlockSize1;
		end = 0;

		hipSetDevice(0);
		checkCudaErrors( hipMemcpy(d_end, &end, sizeof(*d_end), hipMemcpyHostToDevice) );

		hipSetDevice(1);
		checkCudaErrors( hipMemcpy(d_begin, &begin, sizeof(*d_begin), hipMemcpyHostToDevice) );


		for (int iter = 0; iter < maxiter ; iter++) {

			int top = 0;
			int bottom = nb_tasks;

			if(iter == 0 ){
				hipSetDevice(0);
				//setup prin
				hipMemcpyAsync(d_prout0, d_prior0, nVtx*sizeof(*prior), hipMemcpyDeviceToDevice,stream0);
				//hipMemcpyAsync(d_prin0, d_prior0, (medium)*sizeof(*prior), hipMemcpyDeviceToDevice,stream0);


				hipSetDevice(1);
				//setup prin
				hipMemcpyAsync(d_prout1, d_prior1, nVtx*sizeof(*prior), hipMemcpyDeviceToDevice,stream1);


			}else{
				hipSetDevice(1);
				hipMemcpyAsync(d_prin1, d_prout0, (medium)*sizeof(*prior), hipMemcpyDeviceToDevice, stream1);

				hipSetDevice(0);
				hipMemcpyAsync(d_prin0+medium, d_prout1+medium, (nVtx-medium)*sizeof(*prior), hipMemcpyDeviceToDevice, stream0);


				hipSetDevice(0);
				hipMemcpyAsync(d_prout0, d_prior0, (medium)*sizeof(*prior), hipMemcpyDeviceToDevice, stream0);

				hipSetDevice(1);
				hipMemcpyAsync(d_prout1+medium, d_prior1+medium, (nVtx-medium)*sizeof(*prior), hipMemcpyDeviceToDevice, stream1);
			}
			hipSetDevice(0);
			hipDeviceSynchronize();
			hipSetDevice(1);
			hipDeviceSynchronize();
			int iteration = 0;


			//apel des deux fonctions.



			hipSetDevice(0);
			csr_adaptativeGPU0<<< WGSIZE, nThreadPerBlock, mmshared_size, stream0 >>>(d_end, d_begin, d_val0, d_adj0, d_xadj0, d_prin0, d_prout0, d_rowBlocks0, d_a0, d_b0, d_blkSize0, d_blkMultiplier0, d_rows_for_vector0, rowBlockSize1);
			cudaPrintError("after kernel0");

                        hipSetDevice(1);
			csr_adaptativeGPU1<<< WGSIZE, nThreadPerBlock, mmshared_size, stream1 >>>(d_end, d_begin, d_val1, d_adj1, d_xadj1, d_prin1, d_prout1, d_rowBlocks1, d_a1, d_b1, d_blkSize1, d_blkMultiplier1, d_rows_for_vector1, rowBlockSize1);


			cudaPrintError("after kernel1");
			//hipDeviceSynchronize();
			checkCudaErrors( hipStreamSynchronize(stream1));


			hipSetDevice(0);
			//hipDeviceSynchronize();	
                        checkCudaErrors( hipStreamSynchronize(stream0));

		//	util::timestamp stop2;
		//	cout << " totaltime="<< stop2 - start << endl;
			

//			cout << "medium=" << medium << endl;
			//compute epsilon
			//using prin to compute epsilon
			hipSetDevice(0);
			hipblasSetStream(cublasHandle0, stream0);

			hipblasSaxpy (cublasHandle0, medium, d_epsalpha0, d_prout0, 1, d_prin0, 1); // d_prin = d_prout*-1 + d_prin

			hipblasSasum (cublasHandle0, medium, d_prin0, 1, d_eps0);

			hipMemcpyAsync(h_eps0, d_eps0, sizeof(*d_eps0), hipMemcpyDeviceToHost, stream0);

			//      hipMemcpyAsync(d_prin0, d_prout0, nVtx*sizeof(*prout), hipMemcpyDeviceToDevice, stream0);//prepare prin for next iteration


			//compute epsilon
			//using prin to compute epsilon
			hipSetDevice(1);
			hipblasSetStream(cublasHandle1, stream1);
			hipblasSaxpy (cublasHandle1, (nVtx-medium), d_epsalpha1, d_prout1+medium, 1, d_prin1+medium, 1); // d_prin = d_prout*-1 + d_prin

			hipblasSasum(cublasHandle1, nVtx-medium, d_prin1+medium, 1, d_eps1);

			hipMemcpyAsync(h_eps1, d_eps1, sizeof(*h_eps1), hipMemcpyDeviceToHost, stream1);
	//		hipSetDevice(1);
	//		hipMemcpyAsync(d_prin1+medium, d_prout1+medium, (nVtx-medium)*sizeof(*prout), hipMemcpyDeviceToDevice,stream1);//prepare prin for next iteration

	//		hipSetDevice(0);
	//		hipMemcpyAsync(d_prin0, d_prout0, (medium)*sizeof(*prout), hipMemcpyDeviceToDevice, stream0);//prepare prin for next iteration

			hipSetDevice(1);
			checkCudaErrors( hipStreamSynchronize(stream1));


			hipSetDevice(0);
			checkCudaErrors( hipStreamSynchronize(stream0));

			//stopping condition
	//		if (*h_eps0 +*h_eps1 < 0) // deactivited for testing purposes
	//			iter = maxiter;

			std::cerr<<*h_eps0+*h_eps1<< " " ;

		}

		hipSetDevice(0);
		checkCudaErrors(hipMemcpy(&end, d_end, sizeof(*d_end), hipMemcpyDeviceToHost));
		hipSetDevice(1);
		checkCudaErrors(hipMemcpy(&begin, d_begin, sizeof(*d_begin), hipMemcpyDeviceToHost));


		int rowEnd = ((rowBlocks[end] >> (64-32)) & ((1UL << 32) - 1UL));
		int rowBegin = ((rowBlocks[begin] >> (64-32)) & ((1UL << 32) - 1UL));


		hipSetDevice(0);
		checkCudaErrors(hipMemcpy(prout, d_prout0, 1*sizeof(*prout), hipMemcpyDeviceToHost));
		//hipSetDevice(1);
		//checkCudaErrors(hipMemcpy(prout+medium, d_prout1+medium, (nVtx-medium)*sizeof(*prout), hipMemcpyDeviceToHost));

		std::cerr<< " d_begin=" <<begin << "d_end=" <<  end << endl;
		std::cerr<< " row_begin=" << rowBegin << "row_end=" <<  rowEnd << endl;
		for(int i=0; i<1; i++)
		{
			std::cerr.precision(10);
			std::cerr<<"PR["<< i<< "]="<<prout[i]<<std::endl;
		}




		if (TRY >= THROW_AWAY)
		{
			util::timestamp stop;
			totaltime += stop - start;
			cout << "ws totaltime="<< stop - start << endl;
		}

	}



	hipSetDevice(0);
	hipDeviceReset();
	hipSetDevice(1);
	hipDeviceReset();


	delete[] prin_;


	{
		std::stringstream ss;
		ss<<"part1V: "<< medium <<" part1E: "<<xadj[medium+1]
			<<" part2V: "<<nVtx-(medium)<<" part2E: "<< xadj[nVtx] - xadj[medium+1];
		out = ss.str();
	}




	return 0;
}



