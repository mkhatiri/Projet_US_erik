#include "main-pr.hpp"

#define THROW_AWAY 0 
#include "Padded2DArray.hpp"
#include <omp.h>
#include "memutils.hpp"
#include <cmath>

//#define SHOWLOADBALANCE
#include "logged_array.hpp"

//#define LOG

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <hipblas.h>
#include "hip/hip_runtime_api.h"
#include "LightSpMV_interface.hpp"
#include "streamUtils.hpp"
#include "tbb/concurrent_queue.h"
#include "math.h"
#include "streamUtils.hpp"
#include "tbb/concurrent_queue.h"



	template <typename VertexType, typename EdgeType, typename Scalar>
int main_pr(VertexType nVtx, EdgeType* xadj_, VertexType *adj_, Scalar* val_, Scalar *prior_, Scalar* pr_,
		Scalar lambda,
		int nTry, //algo parameter
		util::timestamp& totaltime, std::string& out
	   )
{

	int subsize = 0;
	int nb_blocks = 0;
	int stream_number = 0;

	{
		char* str = getenv ("NBBLOCK");
		if (str) {
			std::stringstream ss (str);
			ss>>nb_blocks;
			if (!ss)
				std::cerr<<"NBBLOCK invalid"<<std::endl;
		}
	}



	{
		char* str = getenv ("SUBSIZE");
		if (str) {
			std::stringstream ss (str);
			ss>>subsize;
			if (!ss)
				std::cerr<<"SUBSIZE invalid"<<std::endl;
		}
	}





	{
		char* str = getenv ("NBSTREAM");
		if (str) {
			std::stringstream ss (str);
			ss>>stream_number;
			if (!ss)
				std::cerr<<"NBSTREAM invalid"<<std::endl;
		}
	}

	if(nb_blocks == 0 && subsize == 0){
		std::cerr<<"SUBSIZE=??? or  NBBLOCK=???"<<std::endl;
		exit(0);
	}

	if(stream_number == 0){
		std::cerr<<"NBSTREAM=???? "<<std::endl;
		exit(0);
	}



	if(subsize == 0){
		int X;
		X = (int) nVtx/(nb_blocks*2) ;
		X = X / 32 ;
		subsize = (X+1) * 32;
	}

		std::cout << "subsize " << subsize <<std::endl;
		list<Task<int,int,float> > *tasks = new  list<Task<int,int,float> >;

		tbb::concurrent_bounded_queue<stream_container<int,int,float>* >* streams = new tbb::concurrent_bounded_queue<stream_container<int,int,float>* >;

		bool coldcache = true;

		util::timestamp start(0,0);

		//cpuside variables  
		Scalar* prin_ = new Scalar[nVtx];
		EdgeType* xadj = xadj_;
		VertexType *adj = adj_;
		Scalar* val = val_;
		Scalar* prior = prior_;
		Scalar* prin = prin_;
		Scalar* prout = pr_;
		Scalar alpha = lambda;
		Scalar beta = 1-lambda;


		Scalar alpha1 = lambda;
		Scalar beta1 = 1-lambda;
		Scalar epsalpha = -1;

		Scalar *h_eps0;
		Scalar *h_eps1;


		//cuda side variable
		EdgeType* d_xadj0 ;
		VertexType *d_adj0 ;
		Scalar* d_val0 ;
		Scalar* d_prior0 ;
		Scalar* d_prin0 ;
		Scalar* d_prout0 ;
		Scalar *d_alpha0;
		Scalar *d_beta0;
		Scalar *d_epsalpha0;
		Scalar *d_eps0;

		EdgeType* d_xadj1 ;
		VertexType *d_adj1 ;
		Scalar* d_val1 ;
		Scalar* d_prior1 ;
		Scalar* d_prin1 ;
		Scalar* d_prout1 ;
		Scalar *d_alpha1;
		Scalar *d_beta1;

		Scalar *d_epsalpha1;
		Scalar *d_eps1;

		/* Get handle to the CUBLAS context */
		hipSetDevice(0);
		hipblasHandle_t cublasHandle0 = 0;
		hipblasStatus_t cublasStatus0;
		cublasStatus0 = hipblasCreate(&cublasHandle0);
		hipblasSetPointerMode(cublasHandle0, HIPBLAS_POINTER_MODE_DEVICE);

		checkCudaErrors(hipSetDevice(1));
		hipblasHandle_t cublasHandle1 = 0;
		hipblasStatus_t cublasStatus1;
		cublasStatus1 = hipblasCreate(&cublasHandle1);
		hipblasSetPointerMode(cublasHandle1, HIPBLAS_POINTER_MODE_DEVICE);

		/* Get handle to the CUSPARSE context */
		hipSetDevice(0);

		hipsparseHandle_t cusparseHandle0 = 0;
		hipsparseStatus_t cusparseStatus0;
		cusparseStatus0 = hipsparseCreate(&cusparseHandle0);

		hipsparseMatDescr_t descr0 = 0;
		cusparseStatus0 = hipsparseCreateMatDescr(&descr0);

		hipsparseSetMatType(descr0,HIPSPARSE_MATRIX_TYPE_GENERAL);
		hipsparseSetMatIndexBase(descr0,HIPSPARSE_INDEX_BASE_ZERO);


		hipSetDevice(1);

		hipsparseHandle_t cusparseHandle1 = 0;
		hipsparseStatus_t cusparseStatus1;
		cusparseStatus1 = hipsparseCreate(&cusparseHandle1);

		hipsparseMatDescr_t descr1 = 0;
		cusparseStatus1 = hipsparseCreateMatDescr(&descr1);

		hipsparseSetMatType(descr1,HIPSPARSE_MATRIX_TYPE_GENERAL);
		hipsparseSetMatIndexBase(descr1,HIPSPARSE_INDEX_BASE_ZERO);


		//cuda stream

		hipSetDevice(0);

		hipStream_t stream0;
		hipStreamCreate(&stream0);

		hipSetDevice(1);

		hipStream_t stream1;
		hipStreamCreate(&stream1);
//i		vector<lightSpMVCSRKernel> lspmv1(stream_number)  ;

		//memalloc

		hipSetDevice(0);

		checkCudaErrors( hipMalloc((void**)&d_xadj0, (nVtx+1)*sizeof(*xadj)) );
		checkCudaErrors( hipMalloc((void**)&d_adj0, (xadj[nVtx])*sizeof(*adj)) );
		checkCudaErrors( hipMalloc((void**)&d_val0, (xadj[nVtx])*sizeof(*val)) );
		checkCudaErrors( hipMalloc((void**)&d_prior0, (nVtx*sizeof(*prior))));
		checkCudaErrors( hipMalloc((void**)&d_prin0, (nVtx*sizeof(*prin)) ));
		checkCudaErrors( hipMalloc((void**)&d_prout0, (nVtx*sizeof(*prout)) ));
		checkCudaErrors( hipMalloc((void**)&d_epsalpha0, (sizeof(epsalpha)) ));
		checkCudaErrors( hipHostMalloc((void**)&h_eps0, (sizeof(*h_eps0)) ));
		checkCudaErrors( hipMalloc((void**)&d_eps0, (sizeof(*h_eps0)) ));


		hipSetDevice(1);

		checkCudaErrors( hipMalloc((void**)&d_xadj1, (nVtx+1)*sizeof(*xadj)) );
		checkCudaErrors( hipMalloc((void**)&d_adj1, (xadj[nVtx])*sizeof(*adj)) );
		checkCudaErrors( hipMalloc((void**)&d_val1, (xadj[nVtx])*sizeof(*val)) );
		checkCudaErrors( hipMalloc((void**)&d_prior1, (nVtx*sizeof(*prior))));
		checkCudaErrors( hipMalloc((void**)&d_prin1, (nVtx*sizeof(*prin)) ));
		checkCudaErrors( hipMalloc((void**)&d_prout1, (nVtx*sizeof(*prout)) ));
		checkCudaErrors( hipMalloc((void**)&d_epsalpha1, (sizeof(epsalpha)) ));
		checkCudaErrors( hipHostMalloc((void**)&h_eps1, (sizeof(*h_eps1)) ));
		checkCudaErrors( hipMalloc((void**)&d_eps1, (sizeof(*h_eps1)) ));



		hipSetDevice(0);


		checkCudaErrors( hipMemcpy(d_xadj0, xadj, (nVtx+1)*sizeof(*xadj), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(d_adj0, adj, (xadj[nVtx])*sizeof(*adj), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(d_val0, val, (xadj[nVtx])*sizeof(*val), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(d_prior0, prior, nVtx*sizeof(*prior), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(d_epsalpha0, &epsalpha, sizeof(epsalpha), hipMemcpyHostToDevice) );


		hipSetDevice(1);


		checkCudaErrors( hipMemcpy(d_xadj1, xadj, (nVtx+1)*sizeof(*xadj), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(d_adj1, adj, (xadj[nVtx])*sizeof(*adj), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(d_val1, val, (xadj[nVtx])*sizeof(*val), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(d_prior1, prior, nVtx*sizeof(*prior), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy(d_epsalpha1, &epsalpha, sizeof(epsalpha), hipMemcpyHostToDevice) );

		streams->set_capacity(stream_number*3);
		creat_stream_2gpus<int, int, float>(nVtx, &alpha, &beta, d_val0, d_xadj0, d_adj0, d_prin0, d_prout0, d_val1, d_xadj1, d_adj1, d_prin1, d_prout1, &cusparseHandle0, &cusparseHandle1, &descr0, &descr1, streams, stream_number );


		int nb_tasks =	split_input_to_tasks<int, int, float>(xadj, nVtx, subsize, *tasks);

		//	std::cout << " number of tasks " << nb_tasks << std::endl;


		//	int nb_tasks =	smart_split_input_to_tasks<int, int, float>(xadj, nVtx, subsize, *tasks);

		std::cout << "number-blocks: " << nb_tasks << std::endl;

		int medium;
		for (int TRY=0; TRY<THROW_AWAY+nTry; ++TRY)
		{
                hipSetDevice(0);
		vector<lightSpMVCSRKernel> lspmv0(stream_number)  ;
                hipSetDevice(1);
		vector<lightSpMVCSRKernel> lspmv1(stream_number)  ;

			if (TRY >= THROW_AWAY)
				start = util::timestamp();

			int maxiter = 40;

			nVtx % 2 == 0 ? medium = (nVtx/2) : medium = ((nb_tasks - 1)/2);

			//for GPU0

			hipSetDevice(0);
			//setup prin
			//hipMemcpyAsync(d_prin0, d_prior0, nVtx*sizeof(*prior), hipMemcpyDeviceToDevice,stream0);
			hipMemcpyAsync(d_prin0, d_prior0, (medium)*sizeof(*prior), hipMemcpyDeviceToDevice,stream0);


			hipSetDevice(1);
			//setup prin
			hipMemcpyAsync(d_prin1+medium, d_prior1+medium, (nVtx-medium)*sizeof(*prior), hipMemcpyDeviceToDevice,stream1);


			hipSetDevice(1);
			checkCudaErrors( hipStreamSynchronize(stream1));


			hipSetDevice(0);
			checkCudaErrors( hipStreamSynchronize(stream0));


			for (int iter = 0; iter < maxiter ; ++ iter) {

				int top = 0;
				int bottom = nb_tasks;
				//exchange data	
				hipSetDevice(1);
				hipMemcpyAsync(d_prin1, d_prin0, (medium)*sizeof(*d_prin0), hipMemcpyDeviceToDevice, stream1);

				hipSetDevice(0);
				hipMemcpyAsync(d_prin0+medium, d_prin1+medium, (nVtx-medium)*sizeof(*d_prin0), hipMemcpyDeviceToDevice, stream0); 

				//prout = A prin
				//prout = lambda * prout + (1-lambda) prior
				hipSetDevice(0);
				hipMemcpyAsync(d_prout0, d_prior0, (medium)*sizeof(*prior), hipMemcpyDeviceToDevice, stream0);

				hipSetDevice(1);
				hipMemcpyAsync(d_prout1+medium, d_prior1+medium, (nVtx-medium)*sizeof(*prior), hipMemcpyDeviceToDevice, stream1);

				hipSetDevice(1);
				checkCudaErrors( hipStreamSynchronize(stream1));


				hipSetDevice(0);
				checkCudaErrors( hipStreamSynchronize(stream0));
			
				int iteration = 0;
				while(top < bottom){
					iteration++;
					//std::cout << " while : "<<  std::endl;
					stream_container<int, int, float> *current_stream;
					streams->pop(current_stream);
					if(current_stream->device == 0){
				//		std::cout << "0 top++ : " << top <<std::endl;
						Task <int,int,float> t = get_task<int,int,float>(tasks, top++);
						put_work_on_stream<int,int,float>(current_stream,t);
						hipSetDevice(0);					
						lspmv0[current_stream->id].spmvBLAS(current_stream->m, current_stream->n, current_stream->nnz,
								(uint32_t *)(current_stream->d_xadj+current_stream->RowPtr)
								,(uint32_t *)current_stream->d_adj, current_stream->d_val,
								current_stream->d_prin,
								current_stream->d_prout+current_stream->RowPtr,
								*current_stream->alpha, *current_stream->beta,current_stream->stream);

						checkCudaErrors(hipStreamAddCallback(current_stream->stream, call_back , current_stream , 0));
					}else{
						//	std::cout << "1 bottom-- " << bottom << std::endl;
						Task <int,int,float> t = get_task<int,int,float>(tasks, --bottom);
						put_work_on_stream<int,int,float>(current_stream,t);
						
						hipSetDevice(1);					
						lspmv1[current_stream->id].spmvBLAS(current_stream->m, current_stream->n, current_stream->nnz,
								(uint32_t *)(current_stream->d_xadj+current_stream->RowPtr)
								,(uint32_t *)current_stream->d_adj, current_stream->d_val,
								current_stream->d_prin,
								current_stream->d_prout+current_stream->RowPtr,
								*current_stream->alpha, *current_stream->beta,current_stream->stream);

						checkCudaErrors(hipStreamAddCallback(current_stream->stream, call_back , current_stream , 0));

						medium = current_stream->RowPtr;

					}		
					//for float it is S.
					//does prout = alpha A prin + beta prout
					//std::cout << iteration << " ---  GPU " << current_stream->device << " stream "<<current_stream->id<<" prend nVtx " << current_stream->m  << " and NNZ  "  << current_stream->nnz << std::endl;
				}

				hipSetDevice(0);
				hipDeviceSynchronize();
				hipSetDevice(1);
				hipDeviceSynchronize();			

				//      std::cout << " medium : "<< medium << std::endl;

				//compute epsilon
				//using prin to compute epsilon
				hipSetDevice(0);
				hipblasSetStream(cublasHandle0, stream0);

				hipblasSaxpy (cublasHandle0, medium, d_epsalpha0, d_prout0, 1, d_prin0, 1); // d_prin = d_prout*-1 + d_prin

				hipblasSasum (cublasHandle0, medium, d_prin0, 1, d_eps0);

				hipMemcpyAsync(h_eps0, d_eps0, sizeof(*d_eps0), hipMemcpyDeviceToHost, stream0);

				//	hipMemcpyAsync(d_prin0, d_prout0, nVtx*sizeof(*prout), hipMemcpyDeviceToDevice, stream0);//prepare prin for next iteration


				//compute epsilon
				//using prin to compute epsilon
				hipSetDevice(1);
				hipblasSetStream(cublasHandle1, stream1);
				hipblasSaxpy (cublasHandle1, (nVtx-medium), d_epsalpha1, d_prout1+medium, 1, d_prin1+medium, 1); // d_prin = d_prout*-1 + d_prin

				hipblasSasum(cublasHandle1, nVtx-medium, d_prin1+medium, 1, d_eps1);

				hipMemcpyAsync(h_eps1, d_eps1, sizeof(*h_eps1), hipMemcpyDeviceToHost, stream1);
				hipSetDevice(1);
				hipMemcpyAsync(d_prin1+medium, d_prout1+medium, (nVtx-medium)*sizeof(*prout), hipMemcpyDeviceToDevice,stream1);//prepare prin for next iteration

				hipSetDevice(0);
				hipMemcpyAsync(d_prin0, d_prout0, (medium)*sizeof(*prout), hipMemcpyDeviceToDevice, stream0);//prepare prin for next iteration

				hipSetDevice(1);
				checkCudaErrors( hipStreamSynchronize(stream1));


				hipSetDevice(0);
				checkCudaErrors( hipStreamSynchronize(stream0));


				//stopping condition
				if (*h_eps0 +*h_eps1 < 0) // deactivited for testing purposes
					iter = maxiter;

				std::cerr<<*h_eps0+*h_eps1<<std::endl;

			}

			hipSetDevice(0);

			checkCudaErrors(hipMemcpy(prout, d_prout0, nVtx*sizeof(*prout), hipMemcpyDeviceToHost));

			std::cerr<<"PR[0]="<<prout[0]<<std::endl;

			if (TRY >= THROW_AWAY)
			{
				util::timestamp stop;  
				totaltime += stop - start;
			}

		}



		hipSetDevice(0);
		hipDeviceReset();
		hipSetDevice(1);
		hipDeviceReset();



		delete[] prin_;

		int lastinzero  = 1; 
		{
			std::stringstream ss;
			ss<<"part1V: "<< medium <<" part1E: "<<xadj[medium]
				<<" part2V: "<< nVtx-(medium) <<" part2E: "<< xadj[nVtx] - xadj[medium];
			out = ss.str();
		}


		return 0;
	}



