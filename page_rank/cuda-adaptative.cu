#include "main-pr.hpp"

#define THROW_AWAY 0
#include "Padded2DArray.hpp"
#include <omp.h>
#include "memutils.hpp"
#include <cmath>

//#define SHOWLOADBALANCE
#include "logged_array.hpp"

//#define LOG
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <hipblas.h>
#include "hip/hip_runtime_api.h"
#include "math.h"
#include "tbb/concurrent_queue.h"
#include "math.h"
#include "AdaptativeUtils.hpp"

	template <typename VertexType, typename EdgeType, typename Scalar>
int main_pr(VertexType nVtx, EdgeType* xadj_, VertexType *adj_, Scalar* val_, Scalar *prior_, Scalar* pr_,
		Scalar lambda,
		int nTry, //algo parameter
		util::timestamp& totaltime, std::string& 
	   )
{


	int blk_size = 0;
	int nb_threads = 0;

	{
		char* str = getenv ("NBTHREAD");
		if (str) {
			std::stringstream ss (str);
			ss>>nb_threads;
			if (!ss)
				std::cerr<<"NBTHREAD invalid"<<std::endl;
		}
	}



	{
		char* str = getenv ("BLKSIZE");
		if (str) {
			std::stringstream ss (str);
			ss>>blk_size;
			if (!ss)
				std::cerr<<"SUBSIZE invalid"<<std::endl;
		}
	}


	if(nb_threads == 0 && blk_size == 0){
		std::cerr<<"BLKSIZE=??? or  NBTHREAD=???"<<std::endl;
		exit(0);
	}



	bool coldcache = true;

	util::timestamp start(0,0);

	//cpuside variables  
	Scalar* prin_ = new Scalar[nVtx];
	EdgeType* xadj = xadj_;
	VertexType *adj = adj_;
	Scalar* val = val_;
	Scalar* prior = prior_;
	Scalar* prin = prin_;
	Scalar* prout = pr_;
	Scalar alpha = lambda;
	Scalar beta = 1-lambda;


	//cuda side variable
	EdgeType* d_xadj ;
	VertexType *d_adj ;
	Scalar* d_val ;
	Scalar* d_prior ;
	Scalar* d_prin ;
	Scalar* d_prout ;
	Scalar *d_alpha;
	Scalar *d_beta;

	/* Get handle to the CUBLAS context */
	hipblasHandle_t cublasHandle = 0;
	hipblasStatus_t hipblasStatus_t;
	hipblasStatus_t = hipblasCreate(&cublasHandle);

	/* Get handle to the CUSPARSE context */
	hipsparseHandle_t cusparseHandle = 0;
	hipsparseStatus_t cusparseStatus;
	cusparseStatus = hipsparseCreate(&cusparseHandle);

	hipsparseMatDescr_t descr = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descr);

	hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

	//memalloc

	checkCudaErrors( hipMalloc((void**)&d_xadj, (nVtx+1)*sizeof(*xadj)) );
	checkCudaErrors( hipMalloc((void**)&d_adj, (xadj[nVtx])*sizeof(*adj)) );
	checkCudaErrors( hipMalloc((void**)&d_val, (xadj[nVtx])*sizeof(*val)) );
	checkCudaErrors( hipMalloc((void**)&d_prior, (nVtx*sizeof(*prior))));
	checkCudaErrors( hipMalloc((void**)&d_prin, (nVtx*sizeof(*prin)) ));
	checkCudaErrors( hipMalloc((void**)&d_prout, (nVtx*sizeof(*prout)) ));

	//cpu to gpu copies

	checkCudaErrors( hipMemcpy(d_xadj, xadj, (nVtx+1)*sizeof(*xadj), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_adj, adj, (xadj[nVtx])*sizeof(*adj), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_val, val, (xadj[nVtx])*sizeof(*val), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_prior, prior, nVtx*sizeof(*prior), hipMemcpyHostToDevice) );



	int nRows = nVtx;
	unsigned long* rowBlocks;
	const int nThreadPerBlock = nb_threads; 
	const unsigned int blkSize = blk_size;
	const unsigned int blkMultiplier = 3 ;
	const unsigned int rows_for_vector = 2 ;
	const bool allocate_row_blocks = true;

	//device variable
	unsigned long* d_rowBlocks;
	unsigned int* d_blkSize;
	unsigned int* d_rows_for_vector;
	unsigned int* d_blkMultiplier;
	float* d_a;
	float* d_b;
	int rowBlockSize1;
	int rowBlockSize2;


	//calculer rowBlockSize
	rowBlockSize1 = ComputeRowBlocksSize<int,int>(xadj, nVtx, blkSize, blkMultiplier, rows_for_vector, nThreadPerBlock);
	//cout << "rowBlockSize1 : " << rowBlockSize1 << endl;

	//declarer rowBlocks
	rowBlocks = (unsigned long*) calloc(sizeof(unsigned long),rowBlockSize1);

	//calculer rowBlocks
	ComputeRowBlocks<int,int>( rowBlocks, rowBlockSize2, xadj, nVtx, blkSize, blkMultiplier, rows_for_vector, nThreadPerBlock, allocate_row_blocks);

	//	if(rowBlocks[rowBlockSize1] == 0){
	//		rowBlockSize1--;
	//	}



	//malloc for device variable
	checkCudaErrors( hipMalloc((void**)&d_rowBlocks, (rowBlockSize1*sizeof(unsigned long))));
	checkCudaErrors( hipMalloc((void**)&d_blkSize, 1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_rows_for_vector,1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_blkMultiplier, 1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_a, 1*sizeof(float)));
	checkCudaErrors( hipMalloc((void**)&d_b, 1*sizeof(float)));


	//send data to device
	checkCudaErrors( hipMemcpy(d_rowBlocks, rowBlocks, rowBlockSize1*sizeof(unsigned long), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_blkSize, &blkSize, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_rows_for_vector, &rows_for_vector, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_blkMultiplier, &blkMultiplier, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_a, &alpha, 1*sizeof(Scalar), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_b, &beta, 1*sizeof(Scalar), hipMemcpyHostToDevice) );

	int mmshared_size =  (blkSize) * sizeof(float);



	//	csr_adaptative<<<(rowBlockSize1 + 1) , nThreadPerBlock, size >>>(d_val, d_adj, d_xadj, d_prior, d_prout1, d_rowBlocks, d_a,  d_b, d_blkSize, d_blkMultiplier, d_rows_for_vector, rowBlockSize1);



	for (int TRY=0; TRY<THROW_AWAY+nTry; ++TRY)
	{
		if (TRY >= THROW_AWAY)
			start = util::timestamp();

		for (int iter = 0; iter < 40 ; ++ iter) {

			//setup prin
			if (iter == 0)
				//std::copy (prior, prior+nVtx, prin);
				checkCudaErrors(hipMemcpy(d_prin, d_prior, nVtx*sizeof(*prior), hipMemcpyDeviceToDevice));
			else
				//std::copy (prout, prout+nVtx, prin);
				checkCudaErrors(hipMemcpy(d_prin, d_prout, nVtx*sizeof(*prout), hipMemcpyDeviceToDevice));

			Scalar eps = 0.;

			//prout = A prin
			//prout = lambda * prout + (1-lambda) prior

			checkCudaErrors(hipMemcpy(d_prout, d_prior, nVtx*sizeof(*prior), hipMemcpyDeviceToDevice));


			//for float it is S.
			//does prout = alpha A prin + beta prout
			csr_adaptative<<<(rowBlockSize1 + 1) , nThreadPerBlock, mmshared_size >>>(d_val, d_adj, d_xadj, d_prin, d_prout, d_rowBlocks, d_a,  d_b, d_blkSize, d_blkMultiplier, d_rows_for_vector, rowBlockSize1);


			hipDeviceSynchronize();

			cudaPrintError("after Kernel");

			if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
				std::cerr<<"err"<<std::endl;



			//compute epsilon
			//using prin to compute epsilon
			float epsalpha = -1;
			hipblasStatus_t = hipblasSaxpy (cublasHandle, nVtx, &epsalpha, d_prout, 1, d_prin, 1); // d_prin = d_prout*-1 + d_prin

			if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
				std::cerr<<"err 1"<<std::endl;

			hipblasStatus_t = hipblasSasum(cublasHandle, nVtx, d_prin, 1, &eps);
			if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
				std::cerr<<"err 2"<<std::endl;

			//stopping condition
			if (eps < 0) // deactivited for testing purposes
				iter = 20;

			std::cerr<<eps<<std::endl;

		}

		checkCudaErrors(hipMemcpy(prout, d_prout, nVtx*sizeof(*prout), hipMemcpyDeviceToHost));

		std::cerr<<"PR[0]="<<prout[0]<<std::endl;

		if (TRY >= THROW_AWAY)
		{
			util::timestamp stop;  
			totaltime += stop - start;
		}



		/*    
#ifndef LOG
if (coldcache) {
#pragma omp parallel
{
evict_array_from_cache(adj, xadj[nVtx]*sizeof(*adj));
evict_array_from_cache(xadj, (nVtx+1)*sizeof(*xadj));
evict_array_from_cache(val, xadj[nVtx]*sizeof(*val));
evict_array_from_cache(prior, nVtx*sizeof(*prior));
evict_array_from_cache(prin, nVtx*sizeof(*prin));
evict_array_from_cache(prout, nVtx*sizeof(*prout));

#pragma omp barrier
}
}
#endif
		 */
}


hipFree(d_rowBlocks);
hipFree(d_blkSize);
hipFree(d_rows_for_vector);
hipFree(d_blkMultiplier);
hipFree(d_a);
hipFree(d_b);


#ifdef SHOWLOADBALANCE
std::cout<<"load balance"<<std::endl;
for (int i=0; i< 244; ++i)
std::cout<<count[i]<<std::endl;
#endif

delete[] prin_;



return 0;
}



