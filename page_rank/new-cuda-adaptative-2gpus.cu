#include "main-pr.hpp"

#define THROW_AWAY 0
#include "Padded2DArray.hpp"
#include <omp.h>
#include "memutils.hpp"
#include <cmath>

//#define SHOWLOADBALANCE
#include "logged_array.hpp"

//#define LOG
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <hipblas.h>
#include "hip/hip_runtime_api.h"
#include "math.h"
//#include "streamUtils.hpp"
#include "tbb/concurrent_queue.h"
#include "AdaptativeUtils.hpp"


	template <typename VertexType, typename EdgeType, typename Scalar>
int main_pr(VertexType nVtx, EdgeType* xadj_, VertexType *adj_, Scalar* val_, Scalar *prior_, Scalar* pr_,
		Scalar lambda,
		int nTry, //algo parameter
		util::timestamp& totaltime, std::string& out 
	   )
{
	int nb_blocks = 0;
	int blk_size = 0;
	int nb_threads = 0;

	{
		char* str = getenv ("NBTHREAD");
		if (str) {
			std::stringstream ss (str);
			ss>>nb_threads;
			if (!ss)
				std::cerr<<"NBTHREAD invalid"<<std::endl;
		}
	}


	{
		char* str = getenv ("NBBLOCK");
		if (str) {
			std::stringstream ss (str);
			ss>>nb_blocks;
			if (!ss)
				std::cerr<<"NBBLOCK invalid"<<std::endl;
		}
	}




	{
		char* str = getenv ("BLKSIZE");
		if (str) {
			std::stringstream ss (str);
			ss>>blk_size;
			if (!ss)
				std::cerr<<"SUBSIZE invalid"<<std::endl;
		}
	}


	if(nb_threads == 0 ){
		std::cerr<<" NBTHREAD=??? "<<std::endl;
		exit(0);
	}

	if(blk_size == 0 ){
		std::cerr<<" BLKSIZE=??? "<<std::endl;
		exit(0);  
	}
	if(nb_blocks == 0 ){
		std::cerr<<" NBBLOCK=??? "<<std::endl;
		exit(0);
	}

	bool coldcache = true;

	util::timestamp start(0,0);

	//cpuside variables  
	Scalar* prin_ = new Scalar[nVtx];
	EdgeType* xadj = xadj_;
	VertexType *adj = adj_;
	Scalar* val = val_;
	Scalar* prior = prior_;
	Scalar* prin = prin_;
	Scalar* prout = pr_;
	Scalar alpha = lambda;
	Scalar beta = 1-lambda;

	Scalar alpha1 = lambda;
	Scalar beta1 = 1-lambda;
	Scalar epsalpha = -1;

	Scalar *h_eps0;
	Scalar *h_eps1;


	//cuda side variable
	EdgeType* d_xadj0 ;
	VertexType *d_adj0 ;
	Scalar* d_val0 ;
	Scalar* d_prior0 ;
	Scalar* d_prin0 ;
	Scalar* d_prout0 ;
	Scalar *d_alpha0;
	Scalar *d_beta0;
	Scalar *d_epsalpha0;
	Scalar *d_eps0;

	EdgeType* d_xadj1 ;
	VertexType *d_adj1 ;
	Scalar* d_val1 ;
	Scalar* d_prior1 ;
	Scalar* d_prin1 ;
	Scalar* d_prout1 ;
	Scalar *d_alpha1;
	Scalar *d_beta1;

	Scalar *d_epsalpha1;
	Scalar *d_eps1;

	/* Get handle to the CUBLAS context */
	hipSetDevice(0);
	hipblasHandle_t cublasHandle0 = 0;
	hipblasStatus_t cublasStatus0;
	cublasStatus0 = hipblasCreate(&cublasHandle0);
	hipblasSetPointerMode(cublasHandle0, HIPBLAS_POINTER_MODE_DEVICE);

	checkCudaErrors( hipSetDevice(1));
	hipblasHandle_t cublasHandle1 = 0;
	hipblasStatus_t cublasStatus1;
	cublasStatus1 = hipblasCreate(&cublasHandle1);
	hipblasSetPointerMode(cublasHandle1, HIPBLAS_POINTER_MODE_DEVICE);

	/* Get handle to the CUSPARSE context */
	hipSetDevice(0);

	hipsparseHandle_t cusparseHandle0 = 0;
	hipsparseStatus_t cusparseStatus0;
	cusparseStatus0 = hipsparseCreate(&cusparseHandle0);

	hipsparseMatDescr_t descr0 = 0;
	cusparseStatus0 = hipsparseCreateMatDescr(&descr0);

	hipsparseSetMatType(descr0,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr0,HIPSPARSE_INDEX_BASE_ZERO);


	hipSetDevice(1);

	hipsparseHandle_t cusparseHandle1 = 0;
	hipsparseStatus_t cusparseStatus1;
	cusparseStatus1 = hipsparseCreate(&cusparseHandle1);

	hipsparseMatDescr_t descr1 = 0;
	cusparseStatus1 = hipsparseCreateMatDescr(&descr1);

	hipsparseSetMatType(descr1,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr1,HIPSPARSE_INDEX_BASE_ZERO);


	//cuda stream

	hipSetDevice(0);

	hipStream_t stream0;
	hipStreamCreate(&stream0);

	hipSetDevice(1);

	hipStream_t stream1;
	hipStreamCreate(&stream1);

	//memalloc

	hipSetDevice(0);

	checkCudaErrors( hipMalloc((void**)&d_xadj0, (nVtx+1)*sizeof(*xadj)) );
	checkCudaErrors( hipMalloc((void**)&d_adj0, (xadj[nVtx])*sizeof(*adj)) );
	checkCudaErrors( hipMalloc((void**)&d_val0, (xadj[nVtx])*sizeof(*val)) );
	checkCudaErrors( hipMalloc((void**)&d_prior0, (nVtx*sizeof(*prior))));
	checkCudaErrors( hipMalloc((void**)&d_prin0, (nVtx*sizeof(*prin)) ));
	checkCudaErrors( hipMalloc((void**)&d_prout0, (nVtx*sizeof(*prout)) ));
	checkCudaErrors( hipMalloc((void**)&d_epsalpha0, (sizeof(epsalpha)) ));
	checkCudaErrors( hipHostMalloc((void**)&h_eps0, (sizeof(*h_eps0)) ));
	checkCudaErrors( hipMalloc((void**)&d_eps0, (sizeof(*h_eps0)) ));


	hipSetDevice(1);

	checkCudaErrors( hipMalloc((void**)&d_xadj1, (nVtx+1)*sizeof(*xadj)) );
	checkCudaErrors( hipMalloc((void**)&d_adj1, (xadj[nVtx])*sizeof(*adj)) );
	checkCudaErrors( hipMalloc((void**)&d_val1, (xadj[nVtx])*sizeof(*val)) );
	checkCudaErrors( hipMalloc((void**)&d_prior1, (nVtx*sizeof(*prior))));
	checkCudaErrors( hipMalloc((void**)&d_prin1, (nVtx*sizeof(*prin)) ));
	checkCudaErrors( hipMalloc((void**)&d_prout1, (nVtx*sizeof(*prout)) ));
	checkCudaErrors( hipMalloc((void**)&d_epsalpha1, (sizeof(epsalpha)) ));
	checkCudaErrors( hipHostMalloc((void**)&h_eps1, (sizeof(*h_eps1)) ));
	checkCudaErrors( hipMalloc((void**)&d_eps1, (sizeof(*h_eps1)) ));


	//cpu to gpu copies

	hipSetDevice(0);


	checkCudaErrors( hipMemcpy(d_xadj0, xadj, (nVtx+1)*sizeof(*xadj), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_adj0, adj, (xadj[nVtx])*sizeof(*adj), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_val0, val, (xadj[nVtx])*sizeof(*val), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_prior0, prior, nVtx*sizeof(*prior), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_epsalpha0, &epsalpha, sizeof(epsalpha), hipMemcpyHostToDevice) );


	hipSetDevice(1);


	checkCudaErrors( hipMemcpy(d_xadj1, xadj, (nVtx+1)*sizeof(*xadj), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_adj1, adj, (xadj[nVtx])*sizeof(*adj), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_val1, val, (xadj[nVtx])*sizeof(*val), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_prior1, prior, nVtx*sizeof(*prior), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_epsalpha1, &epsalpha, sizeof(epsalpha), hipMemcpyHostToDevice) );




	int nRows = nVtx;
	unsigned long* rowBlocks;
	const int nThreadPerBlock = nb_threads; 
	const unsigned int blkSize = blk_size; 
	const unsigned int blkMultiplier = 3;
	const unsigned int rows_for_vector = 2; 
	const bool allocate_row_blocks = true;

	//device 0 variable 
	unsigned long* d_rowBlocks0;
	unsigned int* d_blkSize0;
	unsigned int* d_rows_for_vector0;
	unsigned int* d_blkMultiplier0;
	float* d_a0;
	float* d_b0;

	//device 1 variable 
	unsigned long* d_rowBlocks1;
	unsigned int* d_blkSize1;
	unsigned int* d_rows_for_vector1;
	unsigned int* d_blkMultiplier1;
	float* d_a1;
	float* d_b1;

	int rowBlockSize1;
	int rowBlockSize2;


	//calculer rowBlockSize
	rowBlockSize1 = ComputeRowBlocksSize<int,int>(xadj, nVtx, blkSize, blkMultiplier, rows_for_vector, nThreadPerBlock);
	//cout << "rowBlockSize1 : " << rowBlockSize1 << endl;

	//declarer rowBlocks
	rowBlocks = (unsigned long*) calloc(sizeof(unsigned long),rowBlockSize1);

	//calculer rowBlocks
	ComputeRowBlocks<int,int>( rowBlocks, rowBlockSize2, xadj, nVtx, blkSize, blkMultiplier, rows_for_vector, nThreadPerBlock, allocate_row_blocks);
	//cout << "rowBlockSize2 : " << rowBlockSize2 <<endl;

	int end = ((rowBlocks[rowBlockSize1] >> (64-32)) & ((1UL << 32) - 1UL));
	//	cout << " end : " << end <<endl;
	//		if(end == 0){
	//			rowBlockSize1--;
	//		}

	int mediumRowblocks = cutRowBlocks(rowBlocks, rowBlockSize1);
	int part2 = rowBlockSize1 - mediumRowblocks;

	//	int medium =  ((rowBlocks[mediumRowblocks] >> (64-32)) & ((1UL << 32) - 1UL));
	//	end = ((rowBlocks[rowBlockSize1] >> (64-32)) & ((1UL << 32) - 1UL));

	//	cout << "rowBlockSize1 : " << rowBlockSize1 << endl;
	//	cout << "mediumRowBlocks :" << mediumRowblocks << endl;
	//	cout << " - medium : " << medium <<endl;
	//	cout << " - part2 = " << part2 << endl;
	//	cout << " - end : -- > " << end << endl;

	//malloc for device 0 variable
	hipSetDevice(0);
	checkCudaErrors( hipMalloc((void**)&d_rowBlocks0, (rowBlockSize1*sizeof(unsigned long))));
	checkCudaErrors( hipMalloc((void**)&d_blkSize0, 1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_rows_for_vector0,1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_blkMultiplier0, 1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_a0, 1*sizeof(float)));
	checkCudaErrors( hipMalloc((void**)&d_b0, 1*sizeof(float)));

	//malloc for device 1 variable
	hipSetDevice(1);
	checkCudaErrors( hipMalloc((void**)&d_rowBlocks1, (rowBlockSize1*sizeof(unsigned long))));
	checkCudaErrors( hipMalloc((void**)&d_blkSize1, 1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_rows_for_vector1,1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_blkMultiplier1, 1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_a1, 1*sizeof(float)));
	checkCudaErrors( hipMalloc((void**)&d_b1, 1*sizeof(float)));




	//send data to device 0 
	hipSetDevice(0);
	checkCudaErrors( hipMemcpy(d_rowBlocks0, rowBlocks, rowBlockSize1*sizeof(unsigned long), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_blkSize0, &blkSize, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_rows_for_vector0, &rows_for_vector, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_blkMultiplier0, &blkMultiplier, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_a0, &alpha, 1*sizeof(Scalar), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_b0, &beta, 1*sizeof(Scalar), hipMemcpyHostToDevice) );

	//send data to device 1 
	hipSetDevice(1);
	checkCudaErrors( hipMemcpy(d_rowBlocks1, rowBlocks, rowBlockSize1*sizeof(unsigned long), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_blkSize1, &blkSize, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_rows_for_vector1, &rows_for_vector, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_blkMultiplier1, &blkMultiplier, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_a1, &alpha, 1*sizeof(Scalar), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_b1, &beta, 1*sizeof(Scalar), hipMemcpyHostToDevice) );


	// prepar stream config

	list<Task> *tasks = new  list<Task>;

	tbb::concurrent_bounded_queue<stream_container<int,int,float>* >* streams = new tbb::concurrent_bounded_queue<stream_container<int,int,float>* >;

	int mmshared_size =  (blkSize + 1) * sizeof(float);

	// int nb_blocks = 512;
	int stream_number = 2;


	int X, subsize;
	X = (int) rowBlockSize1/(nb_blocks) ;

	if(X % 64 == 0){
		subsize = X;
	}else{  
		X = X / 64 ;
		subsize = (X+1) * 64;
	}


	int xadjPtr1 =  ((rowBlocks[rowBlockSize1] >> (64-32)) & ((1UL << 32) - 1UL));

	cout << "rowBlockSize : "<< rowBlockSize1 << "last row " << xadjPtr1 << endl;
	cout << "subsize : "<< subsize << endl;
	cout << "start creat stream " <<endl;
	creat_stream_2gpus<int, int, float>(d_rowBlocks0, d_a0, d_b0, d_val0, d_xadj0, d_adj0, d_prin0, d_prout0, d_blkSize0, d_rows_for_vector0, d_blkMultiplier0, d_rowBlocks1, d_a1, d_b1, d_val1, d_xadj1, d_adj1, d_prin1, d_prout1, d_blkSize1, d_rows_for_vector1, d_blkMultiplier1, streams, stream_number );
	cout << "end creat stream " <<endl;
	cout << "start split task " <<endl;
	int nb_tasks = split_input_to_tasks(rowBlocks, rowBlockSize1, subsize, *tasks);
	cout << "fin split task " <<endl;
	cout << "nb_tasks " << nb_tasks << endl;





	int size =  (blkSize) * sizeof(float);

	int medium;
	for (int TRY=0; TRY<THROW_AWAY+nTry; ++TRY)
	{
		if (TRY >= THROW_AWAY)
			start = util::timestamp();

		int maxiter = 40;

		medium =  ((rowBlocks[mediumRowblocks] >> (64-32)) & ((1UL << 32) - 1UL));
		//for GPU0

		hipSetDevice(0);
		//setup prin
		//hipMemcpyAsync(d_prin0, d_prior0, nVtx*sizeof(*prior), hipMemcpyDeviceToDevice,stream0);
		hipMemcpyAsync(d_prin0, d_prior0, (medium)*sizeof(*prior), hipMemcpyDeviceToDevice,stream0);


		hipSetDevice(1);
		//setup prin
		hipMemcpyAsync(d_prin1+medium, d_prior1+medium, (nVtx-medium)*sizeof(*prior), hipMemcpyDeviceToDevice,stream1);


		hipSetDevice(1);
		checkCudaErrors( hipStreamSynchronize(stream1));


		hipSetDevice(0);
		checkCudaErrors( hipStreamSynchronize(stream0));


		for (int iter = 0; iter < maxiter ; ++ iter) {

			int top = 0;
			int bottom = nb_tasks;
			hipSetDevice(1);
			hipMemcpyAsync(d_prin1, d_prin0, (medium)*sizeof(*d_prin0), hipMemcpyDeviceToDevice, stream1);

			hipSetDevice(0);
			hipMemcpyAsync(d_prin0+medium, d_prin1+medium, (nVtx-medium)*sizeof(*d_prin0), hipMemcpyDeviceToDevice, stream0);


			hipSetDevice(0);
			hipMemcpyAsync(d_prout0, d_prior0, (medium)*sizeof(*prior), hipMemcpyDeviceToDevice, stream0);

			hipSetDevice(1);
			hipMemcpyAsync(d_prout1+medium, d_prior1+medium, (nVtx-medium)*sizeof(*prior), hipMemcpyDeviceToDevice, stream1);

			hipSetDevice(0);
			hipDeviceSynchronize();
			hipSetDevice(1);
			hipDeviceSynchronize();
			int iteration = 0;

			while(top < bottom){
				iteration++;
				//std::cout << " while : "<<  std::endl;
				stream_container<int, int, float> *current_stream;
				streams->pop(current_stream);
				if(current_stream->device == 0){
					// std::cout << "0 top++ : " << top <<std::endl;
					Task  t = get_task(tasks, top++);
					put_work_on_stream<int,int,float>(current_stream,t);
				}else{
					// std::cout << "1 bottom-- " << bottom << std::endl;
					Task  t = get_task(tasks, --bottom);
					put_work_on_stream<int,int,float>(current_stream,t);
				}
				hipSetDevice(current_stream->device);

				csr_adaptative<<<(current_stream->rowBlockSize + 1 ) , nThreadPerBlock, mmshared_size, current_stream->stream >>>(current_stream->d_val, current_stream->d_adj, current_stream->d_xadj, current_stream->d_prin, current_stream->d_prout, (current_stream->d_rowBlocks + current_stream->rowBlocksPtr ), current_stream->alpha, current_stream->beta, current_stream->d_blkSize, current_stream->d_blkMultiplier, current_stream->d_rows_for_vector, current_stream->rowBlockSize);

				hipStreamAddCallback(current_stream->stream, call_back , current_stream , 0);
				if(current_stream->device == 1)
					medium = ((rowBlocks[current_stream->rowBlocksPtr] >> (64-32)) & ((1UL << 32) - 1UL));;
			}

			hipSetDevice(0);
			hipDeviceSynchronize();
			hipSetDevice(1);
			hipDeviceSynchronize();

			//compute epsilon
			//using prin to compute epsilon
			hipSetDevice(0);
			hipblasSetStream(cublasHandle0, stream0);

			hipblasSaxpy (cublasHandle0, medium, d_epsalpha0, d_prout0, 1, d_prin0, 1); // d_prin = d_prout*-1 + d_prin

			hipblasSasum (cublasHandle0, medium, d_prin0, 1, d_eps0);

			hipMemcpyAsync(h_eps0, d_eps0, sizeof(*d_eps0), hipMemcpyDeviceToHost, stream0);

			//      hipMemcpyAsync(d_prin0, d_prout0, nVtx*sizeof(*prout), hipMemcpyDeviceToDevice, stream0);//prepare prin for next iteration


			//compute epsilon
			//using prin to compute epsilon
			hipSetDevice(1);
			hipblasSetStream(cublasHandle1, stream1);
			hipblasSaxpy (cublasHandle1, (nVtx-medium), d_epsalpha1, d_prout1+medium, 1, d_prin1+medium, 1); // d_prin = d_prout*-1 + d_prin

			hipblasSasum(cublasHandle1, nVtx-medium, d_prin1+medium, 1, d_eps1);

			hipMemcpyAsync(h_eps1, d_eps1, sizeof(*h_eps1), hipMemcpyDeviceToHost, stream1);
			hipSetDevice(1);
			hipMemcpyAsync(d_prin1+medium, d_prout1+medium, (nVtx-medium)*sizeof(*prout), hipMemcpyDeviceToDevice,stream1);//prepare prin for next iteration

			hipSetDevice(0);
			hipMemcpyAsync(d_prin0, d_prout0, (medium)*sizeof(*prout), hipMemcpyDeviceToDevice, stream0);//prepare prin for next iteration

			hipSetDevice(1);
			checkCudaErrors( hipStreamSynchronize(stream1));


			hipSetDevice(0);
			checkCudaErrors( hipStreamSynchronize(stream0));

			//stopping condition
			if (*h_eps0 +*h_eps1 < 0) // deactivited for testing purposes
				iter = maxiter;

			std::cerr<<*h_eps0+*h_eps1<<std::endl;

		}

		hipSetDevice(0);

		checkCudaErrors(hipMemcpy(prout, d_prout0, nVtx*sizeof(*prout), hipMemcpyDeviceToHost));

		std::cerr<<"PR[0]="<<prout[0]<<std::endl;

		if (TRY >= THROW_AWAY)
		{
			util::timestamp stop;
			totaltime += stop - start;
		}

	}



	hipSetDevice(0);
	hipDeviceReset();
	hipSetDevice(1);
	hipDeviceReset();


	delete[] prin_;


	{
		std::stringstream ss;
		ss<<"part1V: "<< medium <<" part1E: "<<xadj[medium+1]
			<<" part2V: "<<nVtx-(medium)<<" part2E: "<< xadj[nVtx] - xadj[medium+1];
		out = ss.str();
	}




	return 0;
}



