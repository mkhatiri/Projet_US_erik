#include "main-pr.hpp"

#define THROW_AWAY 0
#include "Padded2DArray.hpp"
#include <omp.h>
#include "memutils.hpp"
#include <cmath>

//#define SHOWLOADBALANCE
#include "logged_array.hpp"

//#define LOG
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <hipblas.h>
#include "hip/hip_runtime_api.h"
#include "math.h"
//#include "streamUtils.hpp"
#include "tbb/concurrent_queue.h"
#include "math.h"
#include "AdaptativeUtils.hpp"

	template <typename VertexType, typename EdgeType, typename Scalar>
int main_pr(VertexType nVtx, EdgeType* xadj_, VertexType *adj_, Scalar* val_, Scalar *prior_, Scalar* pr_,
		Scalar lambda,
		int nTry, //algo parameter
		util::timestamp& totaltime, std::string& out 
	   )
{

	int blk_size = 0;
	int nb_threads = 0;

	{
		char* str = getenv ("NBTHREAD");
		if (str) {
			std::stringstream ss (str);
			ss>>nb_threads;
			if (!ss)
				std::cerr<<"NBTHREAD invalid"<<std::endl;
		}
	}



	{
		char* str = getenv ("BLKSIZE");
		if (str) {
			std::stringstream ss (str);
			ss>>blk_size;
			if (!ss)
				std::cerr<<"SUBSIZE invalid"<<std::endl;
		}
	}


	if(nb_threads == 0 && blk_size == 0){
		std::cerr<<"BLKSIZE=??? or  NBTHREAD=???"<<std::endl;
		exit(0);
	}
	
	bool coldcache = true;

	util::timestamp start(0,0);

	//cpuside variables  
	Scalar* prin_ = new Scalar[nVtx];
	EdgeType* xadj = xadj_;
	VertexType *adj = adj_;
	Scalar* val = val_;
	Scalar* prior = prior_;
	Scalar* prin = prin_;
	Scalar* prout = pr_;
	Scalar alpha = lambda;
	Scalar beta = 1-lambda;

	Scalar alpha1 = lambda;
	Scalar beta1 = 1-lambda;
	Scalar epsalpha = -1;

	Scalar *h_eps0;
	Scalar *h_eps1;


	//cuda side variable
	EdgeType* d_xadj0 ;
	VertexType *d_adj0 ;
	Scalar* d_val0 ;
	Scalar* d_prior0 ;
	Scalar* d_prin0 ;
	Scalar* d_prout0 ;
	Scalar *d_alpha0;
	Scalar *d_beta0;
	Scalar *d_epsalpha0;
	Scalar *d_eps0;

	EdgeType* d_xadj1 ;
	VertexType *d_adj1 ;
	Scalar* d_val1 ;
	Scalar* d_prior1 ;
	Scalar* d_prin1 ;
	Scalar* d_prout1 ;
	Scalar *d_alpha1;
	Scalar *d_beta1;

	Scalar *d_epsalpha1;
	Scalar *d_eps1;

	/* Get handle to the CUBLAS context */
	hipSetDevice(0);
	hipblasHandle_t cublasHandle0 = 0;
	hipblasStatus_t cublasStatus0;
	cublasStatus0 = hipblasCreate(&cublasHandle0);
	hipblasSetPointerMode(cublasHandle0, HIPBLAS_POINTER_MODE_DEVICE);

	checkCudaErrors( hipSetDevice(1));
	hipblasHandle_t cublasHandle1 = 0;
	hipblasStatus_t cublasStatus1;
	cublasStatus1 = hipblasCreate(&cublasHandle1);
	hipblasSetPointerMode(cublasHandle1, HIPBLAS_POINTER_MODE_DEVICE);

	/* Get handle to the CUSPARSE context */
	hipSetDevice(0);

	hipsparseHandle_t cusparseHandle0 = 0;
	hipsparseStatus_t cusparseStatus0;
	cusparseStatus0 = hipsparseCreate(&cusparseHandle0);

	hipsparseMatDescr_t descr0 = 0;
	cusparseStatus0 = hipsparseCreateMatDescr(&descr0);

	hipsparseSetMatType(descr0,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr0,HIPSPARSE_INDEX_BASE_ZERO);


	hipSetDevice(1);

	hipsparseHandle_t cusparseHandle1 = 0;
	hipsparseStatus_t cusparseStatus1;
	cusparseStatus1 = hipsparseCreate(&cusparseHandle1);

	hipsparseMatDescr_t descr1 = 0;
	cusparseStatus1 = hipsparseCreateMatDescr(&descr1);

	hipsparseSetMatType(descr1,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr1,HIPSPARSE_INDEX_BASE_ZERO);


	//cuda stream

	hipSetDevice(0);

	hipStream_t stream0;
	hipStreamCreate(&stream0);

	hipSetDevice(1);

	hipStream_t stream1;
	hipStreamCreate(&stream1);

	//memalloc

	hipSetDevice(0);

	checkCudaErrors( hipMalloc((void**)&d_xadj0, (nVtx+1)*sizeof(*xadj)) );
	checkCudaErrors( hipMalloc((void**)&d_adj0, (xadj[nVtx])*sizeof(*adj)) );
	checkCudaErrors( hipMalloc((void**)&d_val0, (xadj[nVtx])*sizeof(*val)) );
	checkCudaErrors( hipMalloc((void**)&d_prior0, (nVtx*sizeof(*prior))));
	checkCudaErrors( hipMalloc((void**)&d_prin0, (nVtx*sizeof(*prin)) ));
	checkCudaErrors( hipMalloc((void**)&d_prout0, (nVtx*sizeof(*prout)) ));
	checkCudaErrors( hipMalloc((void**)&d_epsalpha0, (sizeof(epsalpha)) ));
	checkCudaErrors( hipHostMalloc((void**)&h_eps0, (sizeof(*h_eps0)) ));
	checkCudaErrors( hipMalloc((void**)&d_eps0, (sizeof(*h_eps0)) ));


	hipSetDevice(1);

	checkCudaErrors( hipMalloc((void**)&d_xadj1, (nVtx+1)*sizeof(*xadj)) );
	checkCudaErrors( hipMalloc((void**)&d_adj1, (xadj[nVtx])*sizeof(*adj)) );
	checkCudaErrors( hipMalloc((void**)&d_val1, (xadj[nVtx])*sizeof(*val)) );
	checkCudaErrors( hipMalloc((void**)&d_prior1, (nVtx*sizeof(*prior))));
	checkCudaErrors( hipMalloc((void**)&d_prin1, (nVtx*sizeof(*prin)) ));
	checkCudaErrors( hipMalloc((void**)&d_prout1, (nVtx*sizeof(*prout)) ));
	checkCudaErrors( hipMalloc((void**)&d_epsalpha1, (sizeof(epsalpha)) ));
	checkCudaErrors( hipHostMalloc((void**)&h_eps1, (sizeof(*h_eps1)) ));
	checkCudaErrors( hipMalloc((void**)&d_eps1, (sizeof(*h_eps1)) ));


	//cpu to gpu copies

	hipSetDevice(0);


	checkCudaErrors( hipMemcpy(d_xadj0, xadj, (nVtx+1)*sizeof(*xadj), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_adj0, adj, (xadj[nVtx])*sizeof(*adj), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_val0, val, (xadj[nVtx])*sizeof(*val), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_prior0, prior, nVtx*sizeof(*prior), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_epsalpha0, &epsalpha, sizeof(epsalpha), hipMemcpyHostToDevice) );


	hipSetDevice(1);


	checkCudaErrors( hipMemcpy(d_xadj1, xadj, (nVtx+1)*sizeof(*xadj), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_adj1, adj, (xadj[nVtx])*sizeof(*adj), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_val1, val, (xadj[nVtx])*sizeof(*val), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_prior1, prior, nVtx*sizeof(*prior), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_epsalpha1, &epsalpha, sizeof(epsalpha), hipMemcpyHostToDevice) );




	int nRows = nVtx;
	unsigned long* rowBlocks;
	const int nThreadPerBlock = nb_threads; 
	const unsigned int blkSize = blk_size; 
	const unsigned int blkMultiplier = 3;
	const unsigned int rows_for_vector = 1; 
	const bool allocate_row_blocks = true;

	//device 0 variable 
	unsigned long* d_rowBlocks0;
	unsigned int* d_blkSize0;
	unsigned int* d_rows_for_vector0;
	unsigned int* d_blkMultiplier0;
	float* d_a0;
	float* d_b0;

	//device 1 variable 
	unsigned long* d_rowBlocks1;
	unsigned int* d_blkSize1;
	unsigned int* d_rows_for_vector1;
	unsigned int* d_blkMultiplier1;
	float* d_a1;
	float* d_b1;

	int rowBlockSize1;
	int rowBlockSize2;


	//calculer rowBlockSize
	rowBlockSize1 = ComputeRowBlocksSize<int,int>(xadj, nVtx, blkSize, blkMultiplier, rows_for_vector, nThreadPerBlock);
	//cout << "rowBlockSize1 : " << rowBlockSize1 << endl;

	//declarer rowBlocks
	rowBlocks = (unsigned long*) calloc(sizeof(unsigned long),rowBlockSize1);

	//calculer rowBlocks
	ComputeRowBlocks<int,int>( rowBlocks, rowBlockSize2, xadj, nVtx, blkSize, blkMultiplier, rows_for_vector, nThreadPerBlock, allocate_row_blocks);
	//cout << "rowBlockSize2 : " << rowBlockSize2 <<endl;

        int end = ((rowBlocks[rowBlockSize1] >> (64-32)) & ((1UL << 32) - 1UL));
//	cout << " end : " << end <<endl;
//		if(end == 0){
//			rowBlockSize1--;
//		}

	int mediumRowblocks = cutRowBlocks(rowBlocks, rowBlockSize1);
	int part2 = rowBlockSize1 - mediumRowblocks;
	
	int medium =  ((rowBlocks[mediumRowblocks] >> (64-32)) & ((1UL << 32) - 1UL));
	end = ((rowBlocks[rowBlockSize1] >> (64-32)) & ((1UL << 32) - 1UL));
	
//	cout << "rowBlockSize1 : " << rowBlockSize1 << endl;
//	cout << "mediumRowBlocks :" << mediumRowblocks << endl;
//	cout << " - medium : " << medium <<endl;
//	cout << " - part2 = " << part2 << endl;
//	cout << " - end : -- > " << end << endl;

	//malloc for device 0 variable
	hipSetDevice(0);
	checkCudaErrors( hipMalloc((void**)&d_rowBlocks0, (rowBlockSize1*sizeof(unsigned long))));
	checkCudaErrors( hipMalloc((void**)&d_blkSize0, 1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_rows_for_vector0,1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_blkMultiplier0, 1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_a0, 1*sizeof(float)));
	checkCudaErrors( hipMalloc((void**)&d_b0, 1*sizeof(float)));

	//malloc for device 1 variable
	hipSetDevice(1);
	checkCudaErrors( hipMalloc((void**)&d_rowBlocks1, (rowBlockSize1*sizeof(unsigned long))));
	checkCudaErrors( hipMalloc((void**)&d_blkSize1, 1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_rows_for_vector1,1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_blkMultiplier1, 1*sizeof(unsigned int)));
	checkCudaErrors( hipMalloc((void**)&d_a1, 1*sizeof(float)));
	checkCudaErrors( hipMalloc((void**)&d_b1, 1*sizeof(float)));




	//send data to device 0 
	hipSetDevice(0);
	checkCudaErrors( hipMemcpy(d_rowBlocks0, rowBlocks, rowBlockSize1*sizeof(unsigned long), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_blkSize0, &blkSize, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_rows_for_vector0, &rows_for_vector, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_blkMultiplier0, &blkMultiplier, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_a0, &alpha, 1*sizeof(Scalar), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_b0, &beta, 1*sizeof(Scalar), hipMemcpyHostToDevice) );

	//send data to device 1 
	hipSetDevice(1);
	checkCudaErrors( hipMemcpy(d_rowBlocks1, rowBlocks, rowBlockSize1*sizeof(unsigned long), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_blkSize1, &blkSize, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_rows_for_vector1, &rows_for_vector, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_blkMultiplier1, &blkMultiplier, 1*sizeof(unsigned int), hipMemcpyHostToDevice));
	checkCudaErrors( hipMemcpy(d_a1, &alpha, 1*sizeof(Scalar), hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(d_b1, &beta, 1*sizeof(Scalar), hipMemcpyHostToDevice) );


	int size =  (blkSize) * sizeof(float);

//		csr_adaptative<<<(rowBlockSize1 + 1) , nThreadPerBlock, size >>>(d_val, d_adj, d_xadj, d_prior, d_prout1, d_rowBlocks, d_a,  d_b, d_blkSize, d_blkMultiplier, d_rows_for_vector, rowBlockSize1);


/*	hipSetDevice(0);
	csr_adaptative<<<(mediumRowblocks + 1) , nThreadPerBlock, size, stream0 >>>(d_val0, d_adj0, d_xadj0, d_prior0, d_prout0, d_rowBlocks0, d_a0,  d_b0, d_blkSize0, d_blkMultiplier0, d_rows_for_vector0, mediumRowblocks);

	hipSetDevice(1);
	csr_adaptative<<<(part2 + 1) , nThreadPerBlock, size, stream1 >>>(d_val1, d_adj1, d_xadj1, d_prior1, d_prout1, (d_rowBlocks1 + mediumRowblocks) , d_a1,  d_b1, d_blkSize1, d_blkMultiplier1, d_rows_for_vector1, part2);


        hipSetDevice(0);
	checkCudaErrors(hipMemcpy(prout, d_prout0, nVtx*sizeof(*prout), hipMemcpyDeviceToHost));

        for(int i=0; i< nVtx; i++)
                cout << "pr0["<< i <<"] = "<<prout[i]<<endl;
	cout << "   deuxiem ------ "<< endl  ;     
 
	hipSetDevice(1);
	checkCudaErrors(hipMemcpy(prout, d_prout1, nVtx*sizeof(*prout), hipMemcpyDeviceToHost));


	for(int i=0; i< nVtx; i++)
		cout << "pr1["<< i <<"] = "<<prout[i]<<endl;
*/ 	

	for (int TRY=0; TRY<THROW_AWAY+nTry; ++TRY)
	{
		if (TRY >= THROW_AWAY)
			start = util::timestamp();

		int maxiter = 40;


		//for GPU0

		hipSetDevice(0);
		//setup prin
		//hipMemcpyAsync(d_prin0, d_prior0, nVtx*sizeof(*prior), hipMemcpyDeviceToDevice,stream0);
		hipMemcpyAsync(d_prin0, d_prior0, (medium)*sizeof(*prior), hipMemcpyDeviceToDevice,stream0);


		hipSetDevice(1);
		//setup prin
		hipMemcpyAsync(d_prin1+medium, d_prior1+medium, (nVtx-medium)*sizeof(*prior), hipMemcpyDeviceToDevice,stream1);


		hipSetDevice(1);
		checkCudaErrors( hipStreamSynchronize(stream1));


		hipSetDevice(0);
		checkCudaErrors( hipStreamSynchronize(stream0));


		for (int iter = 0; iter < maxiter ; ++ iter) {

			//exchange data	
			hipSetDevice(1);
			hipMemcpyAsync(d_prin1, d_prin0, (medium)*sizeof(*d_prin0), hipMemcpyDeviceToDevice, stream1);//probably incorrect

			hipSetDevice(0);
			hipMemcpyAsync(d_prin0+medium, d_prin1+medium, (nVtx-medium)*sizeof(*d_prin0), hipMemcpyDeviceToDevice, stream0);//probably incorrect


			hipSetDevice(0);
			//prout = A prin
			//prout = lambda * prout + (1-lambda) prior

			hipMemcpyAsync(d_prout0, d_prior0, (medium)*sizeof(*prior), hipMemcpyDeviceToDevice, stream0);



/*		 cusparseStatus0 = hipsparseScsrmv(cusparseHandle0, HIPSPARSE_OPERATION_NON_TRANSPOSE,
					1+lastinzero, nVtx, xadj[1+lastinzero+1], &alpha,
					descr0,
					d_val0, d_xadj0, d_adj0,
					d_prin0, &beta,
					d_prout0);
*/

		csr_adaptative<<<(mediumRowblocks + 1) , nThreadPerBlock, size, stream0 >>>(d_val0, d_adj0, d_xadj0, d_prin0, d_prout0, d_rowBlocks0, d_a0,  d_b0, d_blkSize0, d_blkMultiplier0, d_rows_for_vector0, mediumRowblocks);


			//compute epsilon
			//using prin to compute epsilon

			hipblasSetStream(cublasHandle0, stream0);

			hipblasSaxpy (cublasHandle0, medium, d_epsalpha0, d_prout0, 1, d_prin0, 1); // d_prin = d_prout*-1 + d_prin

			hipblasSasum (cublasHandle0, medium, d_prin0, 1, d_eps0);

			hipMemcpyAsync(h_eps0, d_eps0, sizeof(*d_eps0), hipMemcpyDeviceToHost, stream0);

			//	hipMemcpyAsync(d_prin0, d_prout0, nVtx*sizeof(*prout), hipMemcpyDeviceToDevice, stream0);//prepare prin for next iteration
			hipMemcpyAsync(d_prin0, d_prout0, (medium)*sizeof(*prout), hipMemcpyDeviceToDevice, stream0);//prepare prin for next iteration


			//for GPU1

			hipSetDevice(1);


			//prout = A prin
			//prout = lambda * prout + (1-lambda) prior

			hipMemcpyAsync(d_prout1+medium, d_prior1+medium, (nVtx-medium)*sizeof(*prior), hipMemcpyDeviceToDevice, stream1);


/*			hipsparseScsrmv(cusparseHandle1, HIPSPARSE_OPERATION_NON_TRANSPOSE,
					nVtx-1-lastinzero, nVtx, xadj[nVtx]-xadj[lastinzero+1+1], &alpha1,
					descr1,
					d_val1, d_xadj1+1+lastinzero, d_adj1,
					d_prin1, &beta1,
					d_prout1+1+lastinzero);
*/
			
			csr_adaptative<<<(part2 + 1) , nThreadPerBlock, size, stream1 >>>(d_val1, d_adj1, d_xadj1, d_prin1, d_prout1, (d_rowBlocks1 + mediumRowblocks) , d_a1,  d_b1, d_blkSize1, d_blkMultiplier1, d_rows_for_vector1, part2);


			//using prin to compute epsilon

			hipblasSetStream(cublasHandle1, stream1);


			hipblasSaxpy (cublasHandle1, (nVtx-medium), d_epsalpha1, d_prout1+medium, 1, d_prin1+medium, 1); // d_prin = d_prout*-1 + d_prin

			hipblasSasum(cublasHandle1, nVtx-medium, d_prin1+medium, 1, d_eps1);

			hipMemcpyAsync(h_eps1, d_eps1, sizeof(*h_eps1), hipMemcpyDeviceToHost, stream1);

			hipMemcpyAsync(d_prin1+medium, d_prout1+medium, (nVtx-medium)*sizeof(*prout), hipMemcpyDeviceToDevice,stream1);//prepare prin for next iteration

			hipSetDevice(1);
			checkCudaErrors( hipStreamSynchronize(stream1));


			hipSetDevice(0);
			checkCudaErrors( hipStreamSynchronize(stream0));


			//stopping condition
			if (*h_eps0 +*h_eps1 < 0) // deactivited for testing purposes
				iter = maxiter;

			std::cerr<<*h_eps0+*h_eps1<<std::endl;

		}

		hipSetDevice(0);

		checkCudaErrors(hipMemcpy(prout, d_prout0, nVtx*sizeof(*prout), hipMemcpyDeviceToHost));

		std::cerr<<"PR[0]="<<prout[0]<<std::endl;

		if (TRY >= THROW_AWAY)
		{
			util::timestamp stop;  
			totaltime += stop - start;
		}

	}


	hipStreamDestroy(stream0);

	delete[] prin_;


	{
		std::stringstream ss;
		ss<<"part1V: "<< medium <<" part1E: "<<xadj[medium+1]
			<<" part2V: "<<nVtx-(medium)<<" part2E: "<< xadj[nVtx] - xadj[medium+1];
		out = ss.str();
	}




	return 0;
}



